#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod_slice/cuda.cu
 * Per-element multiplication of a tensor by a broadcasted slice on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/prod_slice/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::prod_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, Scalar alpha_, const T *src,
        T *dst)
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a global function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] mk: Product of m and k
 * @param[in] alpha_: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    if(i0 < m and i1 < n and i2 < k)
    {
        // Pointer to a corresponding fiber of the output array dst
        T *dst_fiber = dst + i1*mk + i0;
        // Value to multiply by the output fiber
        const Y src_val = alpha * Y{src[i1*m+i0]};
        // Update output value
        dst_fiber[i2*m] = T{Y{dst_fiber[i2*m]} * src_val};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src, T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted slice on CUDA
/*! This is a host function that does the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[i,j]
 *
 * @param[in] m: Size of the first mode of src and dst tensors
 * @param[in] n: Size of the last mode of src and dst tensors
 * @param[in] k: Size of the middle mode of dst tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous m-by-n array
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha,
            src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::prod_slice
