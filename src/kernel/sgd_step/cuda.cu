#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sgd_step/cuda.cu
 * SGD with momentum step with buffers on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sgd_step/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sgd_step
{

template<typename T>
static __global__
void cuda_kernel(Index num_elems, Scalar momentum, Scalar lr, Scalar weight_decay, bool nesterov,
        const T *grad, T *velocity, T *p)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    if(i < num_elems)
    {
        // Read values (param+grad) from RAM only once
        Y p_val = static_cast<Y>(p[i]), grad_val = static_cast<Y>(grad[i]);
        if (weight_decay != 0)
        {
            grad_val += weight_decay * p_val;
        }
        // Read velocity from RAM
        Y velocity_val = static_cast<Y>(velocity[i]);
        // Update velocity: velocity = momentum * velocity + lr * grad
        velocity_val = momentum * velocity_val + lr * grad_val;
        // Store updated velocity
        velocity[i] = static_cast<T>(velocity_val);
        // Update parameters
        if (nesterov)
        {
            // Nesterov: p = p - lr * (grad + momentum * velocity)
            Y effective_grad = grad_val + momentum * velocity_val;
            p[i] = static_cast<T>(p_val - lr * effective_grad);
        }
        else
        {
            // Standard momentum: p = p - velocity
            p[i] = static_cast<T>(p_val - velocity_val);
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index num_elems, Scalar momentum, Scalar lr,
        Scalar weight_decay, bool nesterov, const T *grad_, T *velocity_, T *p_)
    noexcept
//! Fused SGD with momentum step operation of buffers
/*!
* @param[in] stream: CUDA stream
* @param[in] num_elems: Number of elements in buffers
* @param[in] momentum: momentum coefficient
* @param[in] lr: learning rate
* @param[in] weight_decay: coefficient for l2 regularizer
* @param[in] nesterov: whether to use Nesterov momentum
* @param[in] grad: Input buffer stored gradient
* @param[inout] velocity: Input buffer stored velocity (momentum buffer)
* @param[inout] p: Input buffers with parameter that are updated in the end
 * */
{
    dim3 blocks((num_elems+255)/256), threads(256);
    using Y = typename T::repr_t;
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(num_elems,
            Y{momentum}, Y{lr}, Y{weight_decay}, nesterov, grad_, velocity_, p_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, bool nesterov, const fp32_t *grad, fp32_t *velocity,
        fp32_t *p)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, bool nesterov, const fp64_t *grad, fp64_t *velocity,
        fp64_t *p)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, bool nesterov, const bf16_t *grad, bf16_t *velocity,
        bf16_t *p)
    noexcept;

template
void cuda<fp16_t>(hipStream_t stream, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, bool nesterov, const fp16_t *grad, fp16_t *velocity,
        fp16_t *p)
    noexcept;

} // namespace nntile::kernel::sgd_step
