#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sgd_step/cuda.cu
 * SGD with momentum step with buffers on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sgd_step/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sgd_step
{

template<typename T>
static __global__
void cuda_kernel(Index num_iter, Index num_elems, Scalar momentum, Scalar lr, Scalar weight_decay, Scalar dampening, bool nesterov,
        const T *grad, T *velocity, T *p)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    if(i < num_elems)
    {
        // Read values (param+grad) from RAM only once
        Y p_val = static_cast<Y>(p[i]), grad_val = static_cast<Y>(grad[i]);
        if (weight_decay != 0)
        {
            grad_val += weight_decay * p_val;
        }
        if (momentum != 0.0)
        {
            Y velocity_val;
            if (num_iter == 1)
            {
                velocity_val = grad_val;
            }
            else
            {
                velocity_val = momentum * static_cast<Y>(velocity[i]) +
                    static_cast<Y>(1.0 - dampening) * grad_val;
            }
            velocity[i] = static_cast<T>(velocity_val);
            if (nesterov)
            {
                grad_val += momentum * velocity_val;
            }
            else
            {
                grad_val = velocity_val;
            }
        }
        p[i] = static_cast<T>(p_val - lr * grad_val);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index num_iter, Index num_elems, Scalar momentum, Scalar lr,
        Scalar weight_decay, Scalar dampening, bool nesterov, const T *grad_, T *velocity_, T *p_)
    noexcept
//! Fused SGD with momentum step operation of buffers
/*!
* @param[in] stream: CUDA stream
* @param[in] num_iter: current iteration number
* @param[in] num_elems: Number of elements in buffers
* @param[in] momentum: momentum coefficient
* @param[in] lr: learning rate
* @param[in] weight_decay: coefficient for l2 regularizer
* @param[in] dampening: dampening coefficient for momentum
* @param[in] nesterov: whether to use Nesterov momentum
* @param[in] grad: Input buffer stored gradient
* @param[inout] velocity: Input buffer stored velocity (momentum buffer)
* @param[inout] p: Input buffers with parameter that are updated in the end
 * */
{
    dim3 blocks((num_elems+255)/256), threads(256);
    using Y = typename T::repr_t;
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(num_iter, num_elems,
            Y{momentum}, Y{lr}, Y{weight_decay}, Y{dampening}, nesterov, grad_, velocity_, p_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index num_iter, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, Scalar dampening, bool nesterov, const fp32_t *grad, fp32_t *velocity,
        fp32_t *p)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index num_iter, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, Scalar dampening, bool nesterov, const fp64_t *grad, fp64_t *velocity,
        fp64_t *p)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index num_iter, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, Scalar dampening, bool nesterov, const bf16_t *grad, bf16_t *velocity,
        bf16_t *p)
    noexcept;

template
void cuda<fp16_t>(hipStream_t stream, Index num_iter, Index num_elems, Scalar momentum,
        Scalar lr, Scalar weight_decay, Scalar dampening, bool nesterov, const fp16_t *grad, fp16_t *velocity,
        fp16_t *p)
    noexcept;

} // namespace nntile::kernel::sgd_step
