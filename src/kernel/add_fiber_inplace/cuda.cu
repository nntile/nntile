#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add_fiber_inplace/cuda.cu
 * Per-element addition of a tensor and a broadcasted fiber on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/add_fiber_inplace/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::add_fiber_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index batch, Scalar alpha_, const T *src,
        Scalar beta_, T *dst)
//! Generic implementation of the add_fiber_inplace operation on CUDA
/*! @copydoc nntile::kernel::add_fiber_inplace::cuda
 * */
{
    Index i2 = threadIdx.x + blockIdx.x*blockDim.x,
          i0 = threadIdx.y + blockIdx.y*blockDim.y,
          i1 = threadIdx.z + blockIdx.z*blockDim.z;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    const Y beta{beta_};
    if(i2 < k and i1 < n and i0 < m)
    {
        for(Index b = 0; b < batch; ++b)
        {
            // Value to add to the output slice
            const Y src_val = alpha * Y{src[i2+b*k]};
            // Output fiber to be updated
            T *dst_fiber = dst + ((i1+b*n)*k+i2)*m;
            // Read value from the output
            T &dst_val = dst_fiber[i0];
            // And update it
            dst_val = T{beta * Y{dst_val} + src_val};
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const T *src, Scalar beta, T *dst)
    noexcept
//! Add a broadcasted fiber into a tensor inplace with optional scaling on CUDA
/*! Performs the following operations:
 *      dst[i,l,j] = beta*dst[i,l,j] + alpha*src[l]
 *
 * This function reads both src and dst even if alpha or beta is zero.
 * If alpha is zero and src[l,b] is NaN, then dst[i,l,j,b] will be NaN.
 * If beta is zero and dst[i,l,j,b] is NaN, then dst[i,l,j,b] will be NaN.
 * If such behaviour is not desired, then in a case of alpha being zero,
 * use nntile::kernel::scale_inplace, and in a case of beta being zero,
 * use nntile::kernel::scale_fiber instead.
 * If both alpha and beta are zero, then use nntile::kernel::clear instead.
 *
 * @see nntile::kernel::scale_inplace
 * @see nntile::kernel::scale_fiber
 * @see nntile::kernel::clear
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensors
 * @param[in] batch: Size of the batch dimension
 * @param[in] alpha: Scalar factor for src
 * @param[in] src: Input contiguous vector with k elements
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(k), 1024), std::min(int(m), 1),
            std::min(int(n), 1));
    dim3 blocks((k+threads.x-1)/threads.x, (m+threads.y-1)/threads.y,
            (n+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, batch, alpha,
            src, beta, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const fp32_t *src, Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const fp64_t *src, Scalar beta, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const bf16_t *src, Scalar beta, bf16_t *dst)
    noexcept;

template
void cuda<fp16_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const fp16_t *src, Scalar beta, fp16_t *dst)
    noexcept;

} // namespace nntile::kernel::add_fiber_inplace
