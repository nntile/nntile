#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sqrt/cuda.cu
 * Sqrt of buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sqrt/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sqrt
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        dst[i] = ::sqrt(src[i]);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src_, T *dst_)
    noexcept
//! Sqrt operation on CUDA
/*
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] src: Input buffer to apply sqrt
 * @params[out] dst: Output buffer to apply sqrt
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto src = reinterpret_cast<const Y *>(src_);
    auto dst = reinterpret_cast<Y *>(dst_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::sqrt
