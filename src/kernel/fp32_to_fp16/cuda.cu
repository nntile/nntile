#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/fp32_to_fp16/cuda.cu
 * Convert fp32_t array into fp16_t array on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/fp32_to_fp16/cuda.hh"
#include <hip/hip_fp16.h>

namespace nntile::kernel::fp32_to_fp16
{

static __global__
void cuda_kernel(Index nelems, const float *src, __half *dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        dst[i] = __float2half(src[i]);
    }
}

void cuda(hipStream_t stream, Index nelems, const fp32_t *src_, fp16_t *dst_)
    noexcept
/*!
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] src: Input array
 * @params[out] dst: Output array
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    const float *src = reinterpret_cast<const float *>(src_);
    __half *dst = reinterpret_cast<__half *>(dst_);
    (cuda_kernel)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

} // namespace nntile::kernel::fp32_to_fp16
