#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/norm_slice/cuda.cu
 * Euclidean norms of fibers into a slice of a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/norm_slice/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::norm_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, Scalar alpha_, const T *src,
        Scalar beta_, T *dst)
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y;
    Index i2_start = threadIdx.z, i2_step = blockDim.z;
    using Y = typename T::repr_t;
    const Y beta{beta_};
    const Y alpha{alpha_};
    using Z = typename CUDAComputeType<T>::value;
    constexpr Y zero{0.};
    if(i0 < m and i1 < n)
    {
        // Pointer to a corresponding fiber of the source array src
        const T *src_fiber = src + i1*mk + i0;
        // Init sum over the fiber
        Y sum = zero;
        // Cycle over fiber elements and accumulate the sum
        for(Index i2 = i2_start; i2 < k; i2 += i2_step)
        {
            sum = ::hypot(sum, Y{src_fiber[i2*m]});
        }
        volatile __shared__ Y block_max[64];
        __shared__ Y block_sum[64];
        if(i2_start == 0)
        {
            block_max[threadIdx.x+blockDim.x*threadIdx.y] = sum;
            block_sum[threadIdx.x+blockDim.x*threadIdx.y] = zero;
        }
        __syncthreads();
        while(block_max[threadIdx.x+blockDim.x*threadIdx.y] < sum)
        {
            block_max[threadIdx.x+blockDim.x*threadIdx.y] = sum;
        }
        __syncthreads();
        if(block_max[threadIdx.x+blockDim.x*threadIdx.y] > 0)
        {
            sum /= block_max[threadIdx.x+blockDim.x*threadIdx.y];
            atomicAdd(&block_sum[threadIdx.x+blockDim.x*threadIdx.y], sum*sum);
            __syncthreads();
            // Update output value
            if(i2_start == 0)
            {
                // Output value
                T &result = dst[i1*m+i0];
                sum = block_max[threadIdx.x+blockDim.x*threadIdx.y];
                sum *= ::sqrt(block_sum[threadIdx.x+blockDim.x*threadIdx.y]);
                if(beta == zero)
                {
                    result = T{::fabs(alpha) * sum};
                }
                else
                {
                    result = T{::hypot(beta*Y{result}, alpha*sum)};
                }
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src, Scalar beta, T *dst)
    noexcept
//! Euclidean norms over fibers along middle axis into a slice of a tensor
/*! For a provided m-by-k-by-n input array src compute norms of fibers
 * along second axis with k elements, resulting in m-by-n output array-slice
 * dst.
 * Mnemonically, the following operations are performed:
 *      dst[i,j] = hypot(beta*dst[i,j], alpha*norm(src[i,:,j]))
 *
 * @param[in] m: Size of the first mode of src and dst arrays
 * @param[in] n: Size of the last mode of src and dst arrays
 * @param[in] k: Size of the middle mode of src array
 * @param[in] alpha: Scaling factor for src
 * @param[in] src_: Input contiguous m-by-k-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst_: Input and output contiguous m-by-n array, that
 *      accumulates norms along middle axis.
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y, 1);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha,
            src, beta, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src, Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src, Scalar beta, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src, Scalar beta, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::norm_slice
