#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/gelutanh_inplace/cuda.cu
 * Approximate GeLU operation on CUDA based on tanh function
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/gelutanh_inplace/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::gelutanh_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    using Y = typename T::repr_t;
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    // Constants
    constexpr Y pi = 3.141592653589793238462643383279502884L,
        one = 1, f1 = Y{0.044715};
    // Square root is not constexpr by standard, proceed with a static const
    const Y sqrt_pi = sqrt(pi), sqrt_2 = sqrt(Y{2}),
        f2 = sqrt_2/sqrt_pi, f3 = -Y{2}*f2, f4 = f3*f1;
    if(i < nelems)
    {
        Y z = static_cast<Y>(data[i]);
        Y y = z * (f3 + f4*z*z);
        data[i] = T{z / (one+::exp(y))};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Approximate GeLU operation on CUDA
/*! Applies the following approximation of the GeLU function:
 * GeLU(z) \approx 0.5z(1+tanh(sqrt(2/pi)(z+0.044715z^3))),
 * which is actually implemented as
 * GeLU(z) \approx z/(1+exp(-2sqrt(2/pi)z(1+0.044715z^2)))
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data_: Buffer to apply GeLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, bf16_t *data)
    noexcept;

} // namespace nntile::kernel::gelutanh_inplace
