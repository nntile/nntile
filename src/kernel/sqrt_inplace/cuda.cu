#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sqrt_inplace/cuda.cu
 * Inplace sqrt of buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sqrt_inplace/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sqrt_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    using Y = typename T::repr_t;
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        data[i] = T{::sqrt(static_cast<Y>(data[i]))};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace sqrt of buffer
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[inout] data_: Input buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, bf16_t *data)
    noexcept;

} // namespace nntile::kernel::sqrt_inplace
