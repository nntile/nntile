#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sqrt_inplace/cuda.cu
 * Inplace sqrt of buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sqrt_inplace/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sqrt_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        data[i] = ::sqrt(data[i]);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data_)
    noexcept
//! Inplace sqrt of buffer
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[inout] data_: Input buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto data = reinterpret_cast<Y *>(data_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace nntile::kernel::sqrt_inplace
