#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod_fiber_inplace/cuda.cu
 * Per-element multiplication of a tensor by a broadcasted fiber on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/prod_fiber_inplace/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::prod_fiber_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Scalar alpha_, const T *src, T *dst)
//! Per-element product of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[l]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src: Input contiguous vector with k elements
 * @param[inout] dst: Input and output contiguous m-by-k-by-n array
 * */
{
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    if(i0 < m and i1 < n and i2 < k)
    {
        const Y src_val = alpha * Y{src[i2]};
        // Output fiber to be updated
        T *dst_fiber = dst + (i1*k+i2)*m;
        // Update output value
        dst_fiber[i0] = static_cast<T>(src_val * Y{dst_fiber[i0]});
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src, T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = alpha * dst[i,l,j] * src[l]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src_: Input contiguous vector with k elements
 * @param[inout] dst_: Input and output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, alpha, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src, bf16_t *dst)
    noexcept;

template
void cuda<fp16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp16_t *src, fp16_t *dst)
    noexcept;

} // namespace nntile::kernel::prod_fiber_inplace
