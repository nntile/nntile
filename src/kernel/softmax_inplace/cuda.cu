#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/softmax_inplace/cuda.cu
 * Inplace softmax operation on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/softmax_inplace/cuda.hh"
#include <stdio.h>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::softmax_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index m_per_block, Index n, Index n_per_block,
        Index k, const T *maxsumexp, Scalar alpha, T *dst)
{
    Index i0_block = blockIdx.y, i1_block = blockIdx.z,
          i2_start = threadIdx.x, i2_step = blockDim.x;
    using Y = typename T::repr_t;
    constexpr Y zero{0.0};

    for(Index i0 = i0_block*m_per_block;
            i0 < (i0_block+1)*m_per_block and i0 < m; ++i0)
    {
        for(Index i1 = i1_block*n_per_block;
                i1 < (i1_block+1)*n_per_block and i1 < n; ++i1)
        {
            Index dst_offset = i1*k*m + i0;
            T *dst_slice = dst + dst_offset;
            // Max and sum of exponents
            __shared__ Y max, sum;
            if(i2_start == 0)
            {
                Index src_offset = m*i1 + i0;
                max = Y{maxsumexp[2*src_offset]};
                sum = Y{maxsumexp[2*src_offset+1]};
            }
            __syncthreads();
            for(Index i2 = i2_start; i2 < k; i2 += i2_step)
            {
                // Value-to-update
                T &val = dst_slice[i2*m];
                // Update value
                if(not ::isinf(Y{val}))
                {
                    val = T{alpha * ::exp(Y{val}-max) / sum};
                }
                else
                {
                    val = T{zero};
                }
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, const T *maxsumexp,
        Scalar alpha, T *dst)
    noexcept
//! softmax of a buffer along middle axis
/*!
 *
 * @param[in] m: Size of the first mode of dst and sumnorm arrays
 * @param[in] n: Size of the last mode of dst and sumnorm arrays
 * @param[in] k: Size of the middle mode of dst array
 * @param[in] maxsumexp: Maximums and sums of exponents of slices
 * @param[in] alpha: Scalar multiplier for the output
 * @param[in] dst: Contiguous output array
 * */
{
    // Source is an m-by-n matrix and destination is an m-by-k-by-n tensor
    // Both source and destination are Fortran-contiguous
    dim3 threads(32, 1, 1), blocks(1, m, n);
    Index m_per_block = 1, n_per_block = 1;
    if(m > 65535)
    {
        m_per_block = (m+65534) / 65535;
        blocks.y = (m+m_per_block-1) / m_per_block;
    }
    if(n > 65535)
    {
        n_per_block = (n+65534) / 65535;
        blocks.z = (n+n_per_block-1) / n_per_block;
    }
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, m_per_block, n,
            n_per_block, k, maxsumexp, alpha, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_t *maxsumexp, Scalar alpha, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp64_t *maxsumexp, Scalar alpha, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k,
        const bf16_t *maxsumexp, Scalar alpha, bf16_t *dst)
    noexcept;

template
void cuda<fp32_fast_tf32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_fast_tf32_t *maxsumexp, Scalar alpha, fp32_fast_tf32_t *dst)
    noexcept;

} // namespace nntile::kernel::softmax_inplace
