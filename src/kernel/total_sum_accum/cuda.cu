#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/total_sum_accum/cuda.cu
 * total_sum_accum operation on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/total_sum_accum/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::total_sum_accum
{

template<typename T>
static __global__
void cuda_kernel(Scalar alpha_, Index n_labels, Index n_outputs, const T *logsumexp,
        const T *src, const Index *labels, T *val_)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    using Z = typename CUDAComputeType<T>::value;
    auto val = reinterpret_cast<Z *>(val_);
    Y alpha = Y{alpha_};
    if(i < n_outputs)
    {
        __shared__ Y block_val;
        if(threadIdx.x == 0)
        {
            block_val = 0;
        }
        atomicAdd(&block_val, Y{logsumexp[i]} - Y{src[labels[i]+i*n_labels]});
        __syncthreads();
        if(threadIdx.x == 0)
        {
            atomicAdd(val, Z{alpha*block_val});
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Scalar alpha, Index n_labels, Index n_outputs,
        const T *logsumexp_, const T *src_, const int64_t *labels_, T *val_)
    noexcept
//! Total sum accumulating from logsumexp and corrected by elements from src
/*! Mnemonically, the following operations are performed:
 * for every i in [0, n_outputs)
 *      val += alpha * (logsumexp[i]-src[labels[i], i]);
 *
 * @param[in] alpha: Scalar multiplier
 * @param[in] n_labels: Number of possible labels
 * @param[in] n_outputs: Number of elements to sum up.
 * @param[in] logsumexp_: Array with logsumexp values of size n_outputs.
 * @param[in] src_: Matrix of size n_labels times n_outputs stored continuously
 *      in Fortran order
 * @param[in] labels_: Array of size n_outputs with correct labels
 * @param[inout] val_: Scalar that accumulates the total sum
 * */
{
    dim3 blocks((n_outputs+255)/256), threads(256);
    using I = typename CUDAComputeType<int64_t>::value;
    auto labels = reinterpret_cast<const I *>(labels_);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(alpha, n_labels,
            n_outputs, logsumexp_, src_, labels, val_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Scalar alpha, Index n_labels,
        Index n_outputs, const fp32_t* logsumexp, const fp32_t* src,
        const int64_t* labels, fp32_t *val)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Scalar alpha, Index n_labels,
        Index n_outputs, const fp64_t* logsumexp, const fp64_t* src,
        const int64_t* labels, fp64_t *val)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Scalar alpha, Index n_labels,
        Index n_outputs, const bf16_t* logsumexp, const bf16_t* src,
        const int64_t* labels, bf16_t *val)
    noexcept;

} // namespace nntile::kernel::total_sum_accum
