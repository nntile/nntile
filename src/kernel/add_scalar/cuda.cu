#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add_scalar/cuda.cu
 * Add scalar operation of buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/add_scalar/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::add_scalar
{

template<typename T>
static __global__
void cuda_kernel(Index num_elements, T alpha, T beta, T* dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < num_elements)
    {
        dst[i] = alpha + beta * dst[i];
    }
}

template<typename T>
void cuda(hipStream_t stream, Index num_elements, Scalar alpha, Scalar beta,
        T *dst_)
    noexcept
//! Add scalar to buffer buffers on CUDA
/*! dst[i] = alpha + beta*dst[i], where alpha and beta are scalars
 *
 * @param[in] num_elements: Size of the src and dst tensors
 * @param[in] alpha: Scalar bias for the dst tensor
 * @param[in] beta: Scalar multiplier for the dst tensor
 * @param[inout] dst_: Destination of the add_scalar operation
 * */
{
    dim3 blocks((num_elements+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto dst = reinterpret_cast<Y *>(dst_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(num_elements, Y{alpha},
            Y{beta}, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index num_elements, Scalar alpha,
        Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index num_elements, Scalar alpha,
        Scalar beta, fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::add_scalar
