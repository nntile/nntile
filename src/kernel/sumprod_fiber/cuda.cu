#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sumprod_fiber/cuda.cu
 * Sums over slices into a fiber of a product of buffers on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sumprod_fiber/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sumprod_fiber
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Scalar alpha_, const T *src1,
        const T *src2, Scalar beta_, T *dst)
//! Sums over slices into a fiber of a product of two tensors on CPU
/*! For two provided m-by-k-by-n input arrays src1 and src2 compute sums of
 * per-element product of corresponding slices along the first and the third
 * axes with m and n elements respectively, resulting in output vector dst with
 * k elements.
 * Mnemonically, the following operations are performed:
 *      dst[l] = beta*dst[l] + alpha*sum_ij(src1[i,l,j] * src2[i,l,j])
 *
 * @param[in] m: Size of the first mode of src1 and src2 tensors
 * @param[in] n: Size of the last mode of src1 and src2 tensors
 * @param[in] k: Size of the middle mode of src1 and src2 tensors and of the
 *      only mode of dst tensor
 * @param[in] alpha_: Scaling factor for src1*src2
 * @param[in] src1: Input contiguous m-by-k-by-n array
 * @param[in] src2: Input contiguous m-by-k-by-n array
 * @param[in] beta_: Scaling factor for dst
 * @param[inout] dst: Output contiguous vector with k elements, that
 *      accumulates sums along the first and the last axes of per-element
 *      products of src1 and src2.
 * */
{
    Index i2 = threadIdx.x + blockIdx.x*blockDim.x;
    Index i0_start = threadIdx.y, i0_step = blockDim.y;
    Index i1_start = threadIdx.z, i1_step = blockDim.z;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    const Y beta{beta_};
    constexpr Y zero{0.0};
    // Init sum of product of the slices
    Y sum = zero;
    if(i2 < k)
    {
        // Cycle over column of src1 and src2
        for(Index i1 = i1_start; i1 < n; i1 += i1_step)
        {
            // Get corresponding fibers of both sources
            const T *src1_fiber = src1 + (i1*k+i2)*m;
            const T *src2_fiber = src2 + (i1*k+i2)*m;
            // Cycle over fibers of inputs
            for(Index i0 = i0_start; i0 < m; i0 += i0_step)
            {
                // Update sum
                sum += Y{src1_fiber[i0]} * Y{src2_fiber[i0]};
            }
        }
    }
    __shared__ Y block_sum[2];
    if(i1_start == 0 and i0_start == 0)
    {
        block_sum[threadIdx.x] = zero;
    }
    __syncthreads();
    atomicAdd(&block_sum[threadIdx.x], sum);
    __syncthreads();
    // Update output value
    if(i1_start == 0 and i0_start == 0 and i2 < k)
    {
        // Output value
        T &result = dst[i2];
        if(beta == zero)
        {
            result = T{alpha * Y{block_sum[threadIdx.x]}};
        }
        else
        {
            result = T{beta * Y{result} + alpha * block_sum[threadIdx.x]};
        }
    }
}

template<typename T, int BLOCK_ROW, int BLOCK_COL, int LOOP>
static __global__
void cuda_kernel_m1(Index n, Index k, Scalar alpha_, const T *src1,
        const T *src2, Scalar beta_, T *dst)
//! Sums over slices into a fiber of a product of two tensors on CPU
/*! For two provided 1-by-k-by-n input arrays src1 and src2 compute sums of
 * per-element product of corresponding slices along the first and the third
 * axes with m and n elements respectively, resulting in output vector dst with
 * k elements.
 * Mnemonically, the following operations are performed:
 *      dst[l] = beta*dst[l] + alpha*sum_j(src1[0,l,j] * src2[0,l,j])
 *
 * @param[in] n: Size of the last mode of src1 and src2 tensors
 * @param[in] k: Size of the middle mode of src1 and src2 tensors and of the
 *      only mode of dst tensor
 * @param[in] alpha_: Scaling factor for src1*src2
 * @param[in] src1: Input contiguous 1-by-k-by-n array
 * @param[in] src2: Input contiguous 1-by-k-by-n array
 * @param[in] beta_: Scaling factor for dst
 * @param[inout] dst: Output contiguous vector with k elements, that
 *      accumulates sums along the first and the last axes of per-element
 *      products of src1 and src2.
 * */
{
    Index src_block_j_end = (n/BLOCK_COL) * BLOCK_COL;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    const Y beta{beta_};
    constexpr int BLOCK_COL_STEP = BLOCK_COL / LOOP;
    __shared__ Y dst_block[BLOCK_ROW][BLOCK_COL_STEP];
    Y dst_val = 0.0;
    Index src_l = threadIdx.x % BLOCK_ROW;
    Index src_j = threadIdx.x / BLOCK_ROW;
    Index src_offset = blockIdx.x*BLOCK_ROW + src_l + src_j*k;
    // Pointer to a corresponding fiber of the input arrays
    if(src_l+blockIdx.x*BLOCK_ROW < k)
    {
        for(Index src_block_j = 0; src_block_j < src_block_j_end;
                src_block_j += BLOCK_COL)
        {
            const T *src1_fiber = src1 + src_offset + src_block_j*k;
            const T *src2_fiber = src2 + src_offset + src_block_j*k;
            for(int c = 0; c < BLOCK_COL; c += BLOCK_COL_STEP)
            {
                Y val1 = static_cast<Y>(src1_fiber[c*k]);
                Y val2 = static_cast<Y>(src2_fiber[c*k]);
                dst_val += val1 * val2;
            }
        }
        // Pointer to a corresponding fiber of the input arrays
        const T *src1_fiber = src1 + src_offset + src_block_j_end*k;
        const T *src2_fiber = src2 + src_offset + src_block_j_end*k;
        for(Index c = 0; c < n-src_block_j_end; c += BLOCK_COL_STEP)
        {
            Y val1 = static_cast<Y>(src1_fiber[c*k]);
            Y val2 = static_cast<Y>(src2_fiber[c*k]);
            dst_val += val1 * val2;
        }
    }
    // Put calculated value into shared memory
    dst_block[src_l][src_j] = alpha * dst_val;
    // Inter-warp reduction
    for(int c = BLOCK_COL_STEP>>1; c > 0; c >>= 1)
    {
        __syncthreads();
        if(src_j < c)
        {
            dst_block[src_l][src_j] += dst_block[src_l][src_j+c];
        }
    }
    // Write output
    if(src_j == 0 and src_l+blockIdx.x*BLOCK_ROW < k)
    {
        if(beta == 0.0)
        {
            dst[blockIdx.x*BLOCK_ROW+src_l] = static_cast<T>(
                    static_cast<Y>(dst_block[src_l][0]));
        }
        else
        {
            dst_val = beta * static_cast<Y>(dst[blockIdx.x*BLOCK_ROW+src_l]);
            dst[blockIdx.x*BLOCK_ROW+src_l] = static_cast<T>(
                    dst_val + dst_block[src_l][0]);
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src1, const T *src2, Scalar beta, T *dst)
    noexcept
//! Sums over slices into a fiber of a product of two tensors on CPU
/*! For two provided m-by-k-by-n input arrays src1 and src2 compute sums of
 * per-element product of corresponding slices along the first and the third
 * axes with m and n elements respectively, resulting in output vector dst with
 * k elements.
 * Mnemonically, the following operations are performed:
 *      dst[l] = beta*dst[l] + alpha*sum_ij(src1[i,l,j] * src2[i,l,j])
 *
 * @param[in] m: Size of the first mode of src1 and src2 tensors
 * @param[in] n: Size of the last mode of src1 and src2 tensors
 * @param[in] k: Size of the middle mode of src1 and src2 tensors and of the
 *      only mode of dst tensor
 * @param[in] alpha: Scaling factor for src1*src2
 * @param[in] src1: Input contiguous m-by-k-by-n array
 * @param[in] src2: Input contiguous m-by-k-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Output contiguous vector with k elements, that
 *      accumulates sums along the first and the last axes of per-element
 *      products of src1 and src2.
 * */
{
    // Both source and destination are Fortran-contiguous
    // Separate case for m==1
    if(m == 1)
    {
        dim3 threads(256);
        dim3 blocks((k+31)/32);
        (cuda_kernel_m1<T, 32, 8, 1>)<<<blocks, threads, 0, stream>>>(n, k,
                alpha, src1, src2, beta, dst);
    }
    else
    {
        dim3 threads(2, std::min(int(m), 32), std::min(int(n), 32));
        dim3 blocks((k+threads.x-1)/threads.x, 1, 1);
        (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, alpha, src1,
                src2, beta, dst);
    }
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src1, const fp32_t *src2, Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src1, const fp64_t *src2, Scalar beta, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src1, const bf16_t *src2, Scalar beta, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::sumprod_fiber
