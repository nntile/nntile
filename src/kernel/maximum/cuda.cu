#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/maximum/cuda.cu
 * Maximum operation on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/maximum/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::maximum
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T* src, T* dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < nelems)
    {
        dst[i] = ::fmax(dst[i], src[i]);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src_, T *dst_)
    noexcept
//! Inplace maximum operation on CUDA
/*! Does the following per-element operation:
 * dst[i] := max(src[i], dst[i])
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] src_: input buffer
 * @params[inout] dst_: buffer for comparison and store maximum
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto src = reinterpret_cast<const Y *>(src_);
    auto dst = reinterpret_cast<Y *>(dst_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t* src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t* src,
        fp64_t *dst)
    noexcept;

} // namespace nntile::kernel::maximum
