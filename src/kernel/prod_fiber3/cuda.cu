#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod_fiber3/cuda.cu
 * Per-element multiplication of a tensor by a broadcasted fiber on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/prod_fiber3/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::prod_fiber3
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Scalar alpha_,
        const T * __restrict__ src1, const T * __restrict__ src2,
        T * __restrict__ dst)
//! Per-element product of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = alpha * src1[l] * src2[i,l,j]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensor
 * @param[in] alpha_: Scalar factor
 * @param[in] src1: Input contiguous vector with k elements
 * @param[in] src2: Input contiguous m-by-k-by-n array
 * @param[out] dst: Output contiguous m-by-k-by-n array
 * */
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y,
          i2 = threadIdx.z + blockIdx.z*blockDim.z;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    if(i0 < m and i1 < n and i2 < k)
    {
        const Y src1_val = alpha * Y{src1[i2]};
        // Input fiber to be used
        const T *src2_fiber = src2 + (i1*k+i2)*m;
        // Output fiber to be updated
        T *dst_fiber = dst + (i1*k+i2)*m;
        // Update output value
        dst_fiber[i0] = T{src1_val * Y{src2_fiber[i0]}};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src1, const T *src2, T *dst)
    noexcept
//! Per-element product of a tensor and a broadcasted fiber on CPU
//! Per-element product of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *      dst[i,l,j] = alpha * src1[l] * src2[i,l,j]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src
 *      tensor
 * @param[in] alpha: Scalar factor
 * @param[in] src1: Input contiguous vector with k elements
 * @param[in] src2: Input contiguous m-by-k-by-n array
 * @param[out] dst: Output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y,
            (k+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, alpha, src1,
            src2, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src1, const fp32_t *src2, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src1, const fp64_t *src2, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src1, const bf16_t *src2, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::prod_fiber3
