#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/silu_forward/cuda.cu
 * Forward SiLU operation on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/silu_forward/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::silu_forward
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    constexpr Y one = Y{1.0};
    Y src_val{0.0};
    if(i < nelems)
    {
        src_val = Y{src[i]};
        dst[i] = T{src_val / (one + ::exp(-src_val))};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src_, T *dst_)
    noexcept
//! Forward SiLU operation on CUDA
/*! Does the following per-element operation:
 * dst[i] = src[i] * sigmoid(src[i])
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[in] src_: Input array
 * @params[out] dst_: Output array
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, src_, dst_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp32_fast_tf32_t>(hipStream_t stream, Index nelems, const fp32_fast_tf32_t *src,
        fp32_fast_tf32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, const bf16_t *src,
        bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::relu_forward
