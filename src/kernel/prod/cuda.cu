#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/prod/cuda.cu
 * Per-element product of two buffers on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/prod/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::prod
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src, T *dst)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    if(i < nelems)
    {
        dst[i] = T{Y{dst[i]} * Y{src[i]}};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src, T *dst)
    noexcept
//! Per-element product of two buffers
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[in] src: Input buffer
 * @param[inout] dst: Input buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, src, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp32_fast_tf32_t>(hipStream_t stream, Index nelems, const fp32_fast_tf32_t *src,
        fp32_fast_tf32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, const bf16_t *src,
        bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::prod
