#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/add_fiber/cuda.cu
 * Per-element addition of a tensor and a broadcasted fiber on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/add_fiber/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::add_fiber
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index batch, Scalar alpha_, const T *src1,
        Scalar beta_, const T *src2, T *dst)
//! Per-element addition of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *	dst[i,l,j,b] = beta*src2[i,l,j,b] + alpha*src1[l,b]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst and src2 tensor and the only mode of src1
 *     tensors
 * @param[in] batch: Size of the batch dimension
 * @param[in] alpha_: Scalar factor for src1
 * @param[in] src1: Input contiguous vector with k elements
 * @param[in] beta_: Scaling factor for src2
 * @param[in] src2: Input contiguous tensor with m*k*n*batch elements
 * @param[inout] dst: Output contiguous m-by-k-by-n array
 * */
{
    Index i2 = threadIdx.x + blockIdx.x*blockDim.x,
          i0 = threadIdx.y + blockIdx.y*blockDim.y,
          i1 = threadIdx.z + blockIdx.z*blockDim.z;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    const Y beta{beta_};
    if(i2 < k and i1 < n and i0 < m)
    {
        for(Index b = 0; b < batch; ++b)
        {
            // Value to add to the output slice
            const Y src1_val = alpha * Y{src1[i2+b*k]};
	    const T *src2_fiber = src2 + ((i1+b*n)*k+i2)*m;
            // Output fiber to be updated
            T *dst_fiber = dst + ((i1+b*n)*k+i2)*m;
            // Read value from the output
            T src2_val = src2_fiber[i0];
            // And update output
            dst_fiber[i0] = T{beta * Y{src2_val} + src1_val};
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const T *src1, Scalar beta, const T *src2, T *dst)
    noexcept
//! Per-element addition of a tensor and a broadcasted fiber on CPU
/*! Performs the following operations:
 *	dst[i,l,j] = beta*src2[i,l,j] + alpha*src1[l]
 *
 * @param[in] m: Size of the first mode of dst tensor
 * @param[in] n: Size of the last mode of dst tensor
 * @param[in] k: Size of the middle mode of dst tensor and the only mode of src1
 *  	tensors
 * @param[in] batch: Size of the batch dimension
 * @param[in] alpha: Scalar factor for src1
 * @param[in] src1: Input contiguous vector with k elements
 * @param[in] beta: Scaling factor for src2
 * @param[in] src2: Input contiguous tensor with m*k*n*batch elements
 * @param[inout] dst: Output contiguous m-by-k-by-n array
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(k), 1024), std::min(int(m), 1),
            std::min(int(n), 1));
    dim3 blocks((k+threads.x-1)/threads.x, (m+threads.y-1)/threads.y,
            (n+threads.z-1)/threads.z);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, batch, alpha,
            src1, beta, src2, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const fp32_t *src1, Scalar beta,
	const fp32_t *src2, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const fp64_t *src1, Scalar beta,
	const fp64_t *src2, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Index batch,
        Scalar alpha, const bf16_t *src1, Scalar beta,
	const bf16_t *src2, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::add_fiber
