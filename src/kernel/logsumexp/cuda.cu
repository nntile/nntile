#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/logsumexp/cuda.cu
 * Logsumexp after computed maxsumexp result of a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/logsumexp/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::logsumexp
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T * __restrict__ maxsumexp,
        T * __restrict__ logsumexp)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    if(i < nelems)
    {
        Y maxsumexp_even = Y{maxsumexp[2*i]};
        Y maxsumexp_odd = Y{maxsumexp[2*i+1]};
        logsumexp[i] = maxsumexp_even + ::log(maxsumexp_odd);
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *maxsumexp_,
        T *logsumexp_)
    noexcept
//! Logsumexp of buffer
/*! One of the buffers serves as output
 *
 * @param[in] nelems: Number of elements in both buffers
 * @param[in] maxsumexp_: Input buffer, result of maxsumexp operation
 * @param[out] logsumexp_: Output buffers that contains output in the end
 * */
{
    dim3 blocks((nelems+31)/32), threads(32);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, maxsumexp_,
            logsumexp_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *maxsumexp,
        fp32_t *logsumexp)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *maxsumexp,
        fp64_t *logsumexp)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, const bf16_t *maxsumexp,
        bf16_t *logsumexp)
    noexcept;

} // namespace nntile::kernel::logsumexp
