#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/transpose/cuda.cu
 * Transpose operation on buffers on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/transpose/cuda.hh"
#include <algorithm>
#include <iostream>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::transpose
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Scalar alpha_, const T *src, T *dst)
//! Transpose buffers on CPU
/*! dst[i,j] = alpha * src[j,i]
 *
 * @param[in] m: Number of rows of src and columns of dst
 * @param[in] n: Number of columns of src and rows of dst
 * @param[in] alpha_: Scalar multiplier
 * @param[in] src: Source tensor
 * @param[out] dst: Destination of the add operation
 * */
{
    Index i = threadIdx.x;
    Index j = threadIdx.y;
    Index griddim_x = (m+blockDim.x-1) / blockDim.x;
    Index block_i = blockIdx.x % griddim_x;
    Index block_j = blockIdx.x / griddim_x;
    Index global_i = i + block_i*blockDim.x;
    Index global_j = j + block_j*blockDim.y;
    using Y = typename T::repr_t;
 
    if(global_i < m and global_j < n)
    {
        const Y alpha{alpha_};
        __shared__ T block[8][9];
        block[i][j] = T{alpha * Y{src[global_i + global_j*m]}};
        Index dst_thread_offset = i + j*blockDim.x;
        Index dst_i = dst_thread_offset % blockDim.y;
        Index dst_j = dst_thread_offset / blockDim.y;
        Index global_dst_i = dst_i + block_j*blockDim.y;
        Index global_dst_j = dst_j + block_i*blockDim.x;
        __syncthreads();
        dst[global_dst_i + global_dst_j*n] = block[dst_j][dst_i];
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Scalar alpha, const T *src,
        T *dst)
    noexcept
//! Transpose buffers on CPU
/*! dst[i,j] = alpha * src[j,i]
 *
 * @param[in] m: Number of rows of src and columns of dst
 * @param[in] n: Number of columns of src and rows of dst
 * @param[in] alpha: Scalar multiplier
 * @param[in] src: Source tensor
 * @param[out] dst: Destination of the add operation
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(8, 8);
    dim3 blocks(((m+threads.x-1)/threads.x) * ((n+threads.y-1)/threads.y));
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, alpha, src, dst);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess)
    {
        std::cerr << "Error in src::kernel::transpose::cuda<T>\n";
        std::cerr << "m=" << m << " n=" << n << "\n";
    }
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const fp32_t* src, fp32_t* dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const fp64_t* src, fp64_t* dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const bf16_t* src, bf16_t* dst)
    noexcept;

} // namespace nntile::kernel::tranpose
