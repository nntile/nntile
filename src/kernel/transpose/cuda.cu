#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/transpose/cuda.cu
 * Transpose operation on buffers on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/transpose/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"
#include "nntile/kernel/scal/cuda.hh"

namespace nntile::kernel::transpose
{

template<typename T, int BLOCK_ROW, int BLOCK_COL, int BLOCK_LOOP>
static __global__
void cuda_kernel(Index m, Index n, Scalar alpha_, const T *src, T *dst)
//! Transpose buffers on CPU
/*! dst[i,j] = alpha * src[j,i]
 *
 * @param[in] m: Number of rows of src and columns of dst
 * @param[in] n: Number of columns of src and rows of dst
 * @param[in] alpha_: Scalar multiplier
 * @param[in] src: Source tensor
 * @param[out] dst: Destination of the add operation
 * */
{
    Index src_i = threadIdx.x % BLOCK_ROW;
    Index src_j = threadIdx.x / BLOCK_ROW;
    Index src_griddim_row = (m+BLOCK_ROW-1) / BLOCK_ROW;
    Index src_block_i = blockIdx.x % src_griddim_row;
    Index src_block_j = blockIdx.x / src_griddim_row;
    Index global_src_i = src_i + src_block_i*BLOCK_ROW;
    Index global_src_j = src_j + src_block_j*BLOCK_COL;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};

    if((src_block_i+1)*BLOCK_ROW <= m and (src_block_j+1)*BLOCK_COL <= n)
    {
        __shared__ T block[BLOCK_ROW][BLOCK_COL+1];
        const T *src_slice = src + global_src_i + global_src_j*m;
        constexpr int BLOCK_COL_STEP = BLOCK_COL / BLOCK_LOOP;
        for(int k = 0; k < BLOCK_COL; k += BLOCK_COL_STEP)
        {
            block[src_i][src_j+k] = T{alpha * Y{src_slice[k*m]}};
        }
        Index dst_i = threadIdx.x % BLOCK_COL;
        Index dst_j = threadIdx.x / BLOCK_COL;
        Index dst_block_i = src_block_j;
        Index dst_block_j = src_block_i;
        Index global_dst_i = dst_i + dst_block_i*BLOCK_COL;
        Index global_dst_j = dst_j + dst_block_j*BLOCK_ROW;
        T *dst_slice = dst + global_dst_i + global_dst_j*n;
        __syncthreads();
        constexpr int BLOCK_ROW_STEP = BLOCK_ROW / BLOCK_LOOP;
        for(int k = 0; k < BLOCK_ROW; k += BLOCK_ROW_STEP)
        {
            dst_slice[k*n] = block[dst_j+k][dst_i];
        }
    }
    else if(global_src_i < m)
    {
        constexpr int BLOCK_COL_STEP = BLOCK_COL / BLOCK_LOOP;
        for(Index new_j = 0; new_j < BLOCK_COL; new_j += BLOCK_COL_STEP)
        {
            if(global_src_j+new_j >= n)
            {
                break;
            }
            dst[global_src_j+new_j+global_src_i*n] =
                 T{alpha * Y{src[global_src_i+(global_src_j+new_j)*m]}};
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Scalar alpha, const T *src,
        T *dst)
    noexcept
//! Transpose buffers on CPU
/*! dst[i,j] = alpha * src[j,i]
 *
 * @param[in] m: Number of rows of src and columns of dst
 * @param[in] n: Number of columns of src and rows of dst
 * @param[in] alpha: Scalar multiplier
 * @param[in] src: Source tensor
 * @param[out] dst: Destination of the add operation
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(256);
    if(m < n)
    {
        if(m == 1)
        {
            scal::cuda<T>(stream, m*n, alpha, src, dst);
        }
        else if(m < 4)
        {
            dim3 blocks(((m+1)/2) * ((n+255)/256));
            (cuda_kernel<T, 2, 256, 2>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(m < 8)
        {
            dim3 blocks(((m+3)/4) * ((n+255)/256));
            (cuda_kernel<T, 4, 256, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(m < 16)
        {
            dim3 blocks(((m+7)/8) * ((n+127)/128));
            (cuda_kernel<T, 8, 128, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(m < 32)
        {
            dim3 blocks(((m+15)/16) * ((n+63)/64));
            (cuda_kernel<T, 16, 64, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else
        {
            dim3 blocks(((m+31)/32) * ((n+31)/32));
            (cuda_kernel<T, 32, 32, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
    }
    else
    {
        if(n == 1)
        {
            scal::cuda<T>(stream, m*n, alpha, src, dst);
        }
        else if(n < 4)
        {
            dim3 blocks(((m+255)/256) * ((n+1)/2));
            (cuda_kernel<T, 256, 2, 2>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(n < 8)
        {
            dim3 blocks(((m+255)/256) * ((n+3)/4));
            (cuda_kernel<T, 256, 4, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(n < 16)
        {
            dim3 blocks(((m+127)/128) * ((n+7)/8));
            (cuda_kernel<T, 128, 8, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else if(n < 32)
        {
            dim3 blocks(((m+63)/64) * ((n+15)/16));
            (cuda_kernel<T, 64, 16, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
        else
        {
            dim3 blocks(((m+31)/32) * ((n+31)/32));
            (cuda_kernel<T, 32, 32, 4>)<<<blocks, threads, 0, stream>>>(m, n,
                    alpha, src, dst);
        }
    }
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const fp32_t* src, fp32_t* dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const fp64_t* src, fp64_t* dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Scalar alpha,
        const bf16_t* src, bf16_t* dst)
    noexcept;

} // namespace nntile::kernel::tranpose
