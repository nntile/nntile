#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/silu_inplace/cuda.cu
 * Inplace SiLU operation on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/silu_inplace/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::silu_inplace
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    constexpr Y one = Y{1.0};
    Y data_val{0.0};
    if(i < nelems)
    {
        data_val = Y{data[i]};
        data[i] = T{data_val / (one + ::exp(-data_val))};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace SiLU operation on CUDA
/*! Does the following per-element operation:
 * data[i] = data[i] * sigmoid(data[i])
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply SiLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, bf16_t *data)
    noexcept;

template
void cuda<fp16_t>(hipStream_t stream, Index nelems, fp16_t *data)
    noexcept;

} // namespace nntile::kernel::silu_inplace