#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/hypot_scalar_inverse/cuda.cu
 * Inverse of a hypot operation of a buffer and a scalar on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/hypot_scalar_inverse/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::hypot_scalar_inverse
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, Scalar eps_, Scalar alpha_, T* dst)
//! Inverse of a hypot of a buffer and a scalar on CUDA
/*! Performs the following operation:
 *      dst[i] = 1.0 / hypot(alpha*dst[i], eps),
 * where alpha and eps are non-zero scalars.
 *
 * @param[in] nelems: Size of the dst tensor
 * @param[in] eps_: Scalar to be added to the hypot result
 * @param[in] alpha_: Scalar multiplier for the dst tensor
 * @param[inout] dst: Destination of the hypot operation
 * */
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    const Y alpha{alpha_};
    const Y eps{eps_};
    if(i < nelems)
    {
        dst[i] = T{Y{1.0} / ::hypot(alpha*Y{dst[i]}, eps)};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, Scalar eps, Scalar alpha, T *dst)
    noexcept
//! Inverse of a hypot of a buffer and a scalar on CUDA
/*! Performs the following operation:
 *      dst[i] = 1.0 / hypot(alpha*dst[i], eps),
 * where alpha and eps are non-zero scalars.
 *
 * @param[in] nelems: Size of the dst tensor
 * @param[in] eps: Scalar to be added to the hypot result
 * @param[in] alpha: Scalar multiplier for the dst tensor
 * @param[inout] dst_: Destination of the hypot operation
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, eps, alpha, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, Scalar eps, Scalar alpha,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, Scalar eps, Scalar alpha,
        fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, Scalar eps, Scalar alpha,
        bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::hypot_scalar_inverse
