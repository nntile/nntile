#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/sum_slice/cuda.cu
 * Sums over fibers into a slice of a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/sum_slice/cuda.hh"
#include <algorithm>
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::sum_slice
{

template<typename T>
static __global__
void cuda_kernel(Index m, Index n, Index k, Index mk, Scalar alpha_, const T *src,
        Scalar beta_, T *dst)
{
    Index i0 = threadIdx.x + blockIdx.x*blockDim.x,
          i1 = threadIdx.y + blockIdx.y*blockDim.y;
    Index i2_start = threadIdx.z, i2_step = blockDim.z;
    using Y = typename T::repr_t;
    Y alpha{alpha_};
    Y beta{beta_};
    constexpr Y zero{0.0};
    if(i0 < m and i1 < n)
    {
        // Pointer to a corresponding fiber of the source array src
        const T *src_fiber = src + i1*mk + i0;
        // Init sum over the fiber
        Y sum{zero};
        // Cycle over fiber elements and accumulate the sum
        for(Index i2 = i2_start; i2 < k; i2 += i2_step)
        {
            sum += Y{src_fiber[i2*m]};
        }
        __shared__ Y block_sum[64];
        if(i2_start == 0)
        {
            block_sum[threadIdx.x+blockDim.x*threadIdx.y] = zero;
        }
        __syncthreads();
        atomicAdd(&block_sum[threadIdx.x+blockDim.x*threadIdx.y], sum);
        __syncthreads();
        // Update output value
        if(i2_start == 0)
        {
            // Output value
            T &result = dst[i1*m+i0];
            sum = block_sum[threadIdx.x+blockDim.x*threadIdx.y];
            if(beta == zero)
            {
                result = T{alpha * sum};
            }
            else
            {
                result = T{beta * Y{result} + alpha * sum};
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const T *src, Scalar beta, T *dst)
    noexcept
//! Sums over fibers along middle axis into a slice of a tensor
/*! For a provided m-by-k-by-n input array computes sums over fibers
 * along second axis with k elements, resulting in m-by-n output slice.
 * Mnemonically, the following operations are performed:
 *      dst[i,j] = beta*dst[i,j] + alpha*sum(src[i,:,j])
 *
 * @param[in] m: Size of the first mode of src and dst arrays
 * @param[in] n: Size of the last mode of src and dst arrays
 * @param[in] k: Size of the middle mode of src array
 * @param[in] alpha: Scaling factor for src
 * @param[in] src: Input contiguous m-by-k-by-n array
 * @param[in] beta: Scaling factor for dst
 * @param[inout] dst: Output contiguous m-by-n array, that accumulates
 *      sums over fibers along middle axis.
 * */
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(std::min(int(m), 8), std::min(int(n), 8),
            std::min(int(k), 16));
    dim3 blocks((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y, 1);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, n, k, m*k, alpha,
            src, beta, dst);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp32_t *src, Scalar beta, fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const fp64_t *src, Scalar beta, fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k, Scalar alpha,
        const bf16_t *src, Scalar beta, bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::sum_slice
