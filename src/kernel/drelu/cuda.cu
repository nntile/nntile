#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/drelu/cuda.cu
 * Derivative of ReLU operation on a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/drelu/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::drelu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T one = 1.0, zero = 0.0;
    if(i < nelems)
    {
        T &z = data[i];
        if(z > zero)
        {
            z = one;
        }
        else
        {
            z = zero;
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data_)
    noexcept
//! Inplace derivative of ReLU operation performed on CUDA
/*! @params[in] nelems: Number of elements in a buffer
 * @params[inout] data_: Buffer to apply derivative of ReLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto data = reinterpret_cast<Y *>(data_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace nntile::kernel::drelu
