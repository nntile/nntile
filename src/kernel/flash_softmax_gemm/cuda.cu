#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/flash_softmax_gemm/cuda.cu
 * CUDA kernel to compute softmax(mask(QK')/sqrt(d))*V using pre-computed maxsumexp
 *
 * @version 1.1.0
 * */

#include <nntile/kernel/flash_softmax_gemm/cuda.hh>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <limits>

namespace nntile::kernel::flash_softmax_gemm
{

/**
 * @brief Copy 2D block from global to shared memory with transposition
 *
 * @tparam T_gmem Type of data in global memory
 * @tparam T_smem Type of data in shared memory
 * @tparam BLOCK_ROWS Number of rows in the input block (columns in output)
 * @tparam BLOCK_COLS Number of columns in the input block (rows in output)
 *
 * @param gmem_ptr Pointer to the start of the block in global memory
 * @param smem_ptr Pointer to the start of the block in shared memory
 * @param gmem_ld Leading dimension of the global memory matrix
 * @param smem_ld Leading dimension of the shared memory matrix
 * @param thread_id Linear thread ID within the block
 * @param block_size Total number of threads in the block
 */
template<typename T_gmem, typename T_smem,
         Index BLOCK_ROWS, Index BLOCK_COLS>
__device__ void gmem_to_smem_transposed(
    const T_gmem* gmem_ptr,
    T_smem* smem_ptr,
    const Index gmem_ld,
    const Index smem_ld,
    const Index thread_id,
    const Index block_size)
{
    // Total number of elements to copy
    constexpr Index TOTAL_ELEMENTS = BLOCK_ROWS * BLOCK_COLS;
    // Make sure total elements is a multiple of 32 (warp size)
    static_assert(TOTAL_ELEMENTS % 32 == 0, "Total elements must be a multiple of 32");

    // Number of elements each thread will copy
    const Index ELEMENTS_PER_THREAD = (TOTAL_ELEMENTS + block_size - 1) / block_size;

    // Each thread copies ELEMENTS_PER_THREAD elements in an interleaved pattern
    for (Index i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        // Calculate linear index for this thread's current element
        const Index linear_idx = thread_id + i * block_size;

        // Skip if beyond the total elements
        if (linear_idx >= TOTAL_ELEMENTS) {
            break;
        }

        // Convert linear index to 2D coordinates in the input matrix
        const Index row_in = linear_idx / BLOCK_COLS;
        const Index col_in = linear_idx % BLOCK_COLS;

        // Transpose: row_out = col_in, col_out = row_in
        if (row_in < BLOCK_ROWS && col_in < BLOCK_COLS) {
            // Read from global memory in row-major order
            const T_gmem val = gmem_ptr[row_in + col_in * gmem_ld];

            // Write to shared memory with transposition (col_in becomes row, row_in becomes col)
            smem_ptr[col_in + row_in * smem_ld] = T_smem{val};
        }
    }
}

/**
 * @brief Vectorized copy 2D block from global to shared memory with transposition
 *
 * @tparam T_gmem Type of data in global memory
 * @tparam T_smem Type of data in shared memory
 * @tparam BLOCK_ROWS Number of rows in the input block (columns in output)
 * @tparam BLOCK_COLS Number of columns in the input block (rows in output)
 *
 * @param gmem_ptr Pointer to the start of the block in global memory
 * @param smem_ptr Pointer to the start of the block in shared memory
 * @param gmem_ld Leading dimension of the global memory matrix
 * @param smem_ld Leading dimension of the shared memory matrix
 * @param thread_id Linear thread ID within the block
 * @param block_size Total number of threads in the block
 */
template<typename T_gmem, typename T_smem,
         Index BLOCK_ROWS, Index BLOCK_COLS>
__device__ void gmem_to_smem_transposed_vec4(
    const T_gmem* __restrict gmem_ptr,
    T_smem* __restrict smem_ptr,
    const Index gmem_ld,
    const Index smem_ld,
    const Index thread_id,
    const Index block_size)
{
    // Ensure block rows is a multiple of 4 for vectorized loads
    static_assert(BLOCK_ROWS % 4 == 0, "Block rows must be a multiple of 4 for vectorized loads");

    // Total number of vector elements to copy (each vector contains 4 elements)
    constexpr Index TOTAL_VEC_ELEMENTS = (BLOCK_ROWS * BLOCK_COLS) / 4;

    // Number of vector elements each thread will copy
    const Index VEC_ELEMENTS_PER_THREAD = (TOTAL_VEC_ELEMENTS + block_size - 1) / block_size;

    // Each thread copies VEC_ELEMENTS_PER_THREAD vector elements
    #pragma unroll
    for (Index i = 0; i < VEC_ELEMENTS_PER_THREAD; ++i) {
        // Calculate linear index for this thread's current vector element
        const Index linear_vec_idx = thread_id + i * block_size;

        // Skip if beyond the total vector elements
        if (linear_vec_idx >= TOTAL_VEC_ELEMENTS) {
            break;
        }

        // Each vector spans 4 rows in the same column
        // const Index col_in = linear_vec_idx / (BLOCK_ROWS / 4);
        // const Index row_vec = linear_vec_idx % (BLOCK_ROWS / 4);
        const Index col_in = linear_vec_idx % BLOCK_COLS;
        const Index row_vec = linear_vec_idx / BLOCK_COLS;
        const Index row_in = row_vec * 4;

        // Only process if within bounds
        // if (col_in < BLOCK_COLS && row_in + 3 < BLOCK_ROWS)
        {
            // Use vectorized load for better memory bandwidth
            // Load 4 consecutive rows from the same column
            float4 vec_val;

            // Manual load of 4 consecutive rows (can't use direct float4 load due to non-contiguous memory)
            vec_val = *reinterpret_cast<const float4*>(&gmem_ptr[row_in + col_in * gmem_ld]);

            // Store with transposition - the column in input becomes row in output
            // Each of the 4 rows becomes a column in the transposed output
            smem_ptr[col_in + (row_in + 0) * smem_ld] = T_smem{vec_val.x};
            smem_ptr[col_in + (row_in + 1) * smem_ld] = T_smem{vec_val.y};
            smem_ptr[col_in + (row_in + 2) * smem_ld] = T_smem{vec_val.z};
            smem_ptr[col_in + (row_in + 3) * smem_ld] = T_smem{vec_val.w};
        }
    }
}

/**
 * @brief Vectorized copy 2D block from global to shared memory with transposition
 *
 * @tparam T_gmem Type of data in global memory
 * @tparam T_smem Type of data in shared memory
 * @tparam BLOCK_ROWS Number of rows in the input block (columns in output)
 * @tparam BLOCK_COLS Number of columns in the input block (rows in output)
 *
 * @param gmem_ptr Pointer to the start of the block in global memory
 * @param smem_ptr Pointer to the start of the block in shared memory
 * @param gmem_ld Leading dimension of the global memory matrix
 * @param smem_ld Leading dimension of the shared memory matrix
 * @param thread_id Linear thread ID within the block
 * @param block_size Total number of threads in the block
 */
template<typename T_gmem, typename T_smem,
         Index BLOCK_ROWS, Index BLOCK_COLS>
__device__ void gmem_to_smem_vec4(
    const T_gmem* __restrict gmem_ptr,
    T_smem* __restrict smem_ptr,
    const Index gmem_ld,
    const Index smem_ld,
    const Index thread_id,
    const Index block_size)
{
    // Ensure block rows is a multiple of 4 for vectorized loads
    static_assert(BLOCK_ROWS % 4 == 0, "Block rows must be a multiple of 4 for vectorized loads");

    // Total number of vector elements to copy (each vector contains 4 elements)
    constexpr Index TOTAL_VEC_ELEMENTS = (BLOCK_ROWS * BLOCK_COLS) / 4;

    // Number of vector elements each thread will copy
    const Index VEC_ELEMENTS_PER_THREAD = (TOTAL_VEC_ELEMENTS + block_size - 1) / block_size;

    // Each thread copies VEC_ELEMENTS_PER_THREAD vector elements
    #pragma unroll
    for (Index i = 0; i < VEC_ELEMENTS_PER_THREAD; ++i) {
        // Calculate linear index for this thread's current vector element
        const Index linear_vec_idx = thread_id + i * block_size;

        // Skip if beyond the total vector elements
        if (linear_vec_idx >= TOTAL_VEC_ELEMENTS) {
            break;
        }

        // Each vector spans 4 rows in the same column
        // const Index col_in = linear_vec_idx / (BLOCK_ROWS / 4);
        // const Index row_vec = linear_vec_idx % (BLOCK_ROWS / 4);
        const Index col_in = linear_vec_idx % BLOCK_COLS;
        const Index row_vec = linear_vec_idx / BLOCK_COLS;
        const Index row_in = row_vec * 4;

        // Only process if within bounds
        // if (col_in < BLOCK_COLS && row_in + 3 < BLOCK_ROWS)
        {
            // Use vectorized load for better memory bandwidth
            // Load 4 consecutive rows from the same column
            float4 vec_val;

            // Manual load of 4 consecutive rows (can't use direct float4 load due to non-contiguous memory)
            vec_val = *reinterpret_cast<const float4*>(&gmem_ptr[row_in + col_in * gmem_ld]);

            // Store with transposition - the column in input becomes row in output
            // Each of the 4 rows becomes a column in the transposed output
            smem_ptr[col_in * smem_ld + row_in + 0] = T_smem{vec_val.x};
            smem_ptr[col_in * smem_ld + row_in + 1] = T_smem{vec_val.y};
            smem_ptr[col_in * smem_ld + row_in + 2] = T_smem{vec_val.z};
            smem_ptr[col_in * smem_ld + row_in + 3] = T_smem{vec_val.w};
        }
    }
}


template<typename T_gmem, typename T_smem, typename T_accum,
         Index HEAD_SIZE, Index Q_BLOCK, Index K_BLOCK,
         Index KQ_HEAD_BLOCK, Index KQ_Q_TILE, Index KQ_K_TILE,
         Index VS_HEAD_BLOCK, Index V_BLOCK, Index VS_HEAD_TILE,
         Index VS_Q_TILE, Index KV_SPLIT, Index NUM_WARPS>
__global__ void flash_softmax_gemm_kernel(
    Index batch, Index seq, T_accum scale,
    const T_gmem * __restrict K, const T_gmem * __restrict Q,
    const bool_t * __restrict mask, const T_gmem * __restrict maxsumexp,
    const T_gmem * __restrict V, T_gmem * __restrict A)
// Every block of warps computes a single (K_BLOCK x Q_BLOCK) block
// of K'Q. Such a block of K'Q is a matrix multiplication of shape
// (HEAD_SIZE x K_BLOCK)^T x (HEAD_SIZE x Q_BLOCK) -> (K_BLOCK x Q_BLOCK).
// It is computed as a sequence of matrix multiplications of shape
// (KQ_HEAD_BLOCK x K_BLOCK)^T x (KQ_HEAD_BLOCK x Q_BLOCK) ->
// (K_BLOCK x Q_BLOCK).
// Therefore, matrix multiplications are done on top of:
// - block K of shape (KQ_HEAD_BLOCK x K_BLOCK),
// - block Q of shape (KQ_HEAD_BLOCK x Q_BLOCK).
// Every warp computes KQ_K_TILE x KQ_Q_TILE tiles of a block of K'Q.
// In addition to computing K'Q, we also compute softmax(mask(K'Q))
// We denote softmax(mask(K'Q)) as S.
//
// Every block of warps computes a HEAD_SIZE x Q_BLOCK block of VS.
// VS is a matrix multiplication of shape
// (HEAD_SIZE x K_BLOCK) x (K_BLOCK x Q_BLOCK) -> HEAD_SIZE x Q_BLOCK.
// Due to possible large head dimension, we split row dimension HEAD_SIZE
// into VS_HEAD_BLOCK blocks. Then, each multiplication
// (VS_HEAD_BLOCK x K_BLOCK) x (K_BLOCK x Q_BLOCK) -> VS_HEAD_BLOCK x Q_BLOCK
// is computed as a sequence of matrix multiplications of shape
// (VS_HEAD_BLOCK x V_BLOCK) x (V_BLOCK x Q_BLOCK) -> VS_HEAD_BLOCK x Q_BLOCK,
// where K_BLOCK is split into V_BLOCK blocks.
// Every warp computes VS_HEAD_TILE x VS_Q_TILE tiles of a block of VS.
{
    using namespace std;

    // Get global indices
    const Index thread_id = threadIdx.x;
    const Index block_size = blockDim.x;
    const Index batch_idx = blockIdx.y;
    const Index q_block_idx = blockIdx.x;
    const Index kv_split_idx = blockIdx.z;

    // Calculate tile ranges
    const Index num_kv_blocks = (seq + K_BLOCK - 1) / K_BLOCK;
    const Index kv_split_num_blocks = (num_kv_blocks + KV_SPLIT - 1) / KV_SPLIT;
    const Index kv_split_size = kv_split_num_blocks * K_BLOCK;
    const Index kv_block_start = kv_split_idx * kv_split_size;
    const Index kv_block_end = ::min(kv_block_start + kv_split_size, seq);

    // Constants for warp-level processing
    constexpr int WARP_SIZE = 32;
    const int warp_id = thread_id / WARP_SIZE;
    const int lane_id = thread_id % WARP_SIZE;

    // Dynamic shared memory allocation
    extern __shared__ char shared_mem[];

    // Calculate offsets for different shared memory arrays
    constexpr int MAX_BLOCK_SIZE = Q_BLOCK * sizeof(T_smem);
    // constexpr int SUMEXP_BLOCK_SIZE = Q_BLOCK * sizeof(T_smem);
    constexpr int LDQ = Q_BLOCK + 4;
    constexpr int Q_BLOCK_SIZE = 2 * KQ_HEAD_BLOCK * LDQ * sizeof(T_smem);
    constexpr int LDK = K_BLOCK + 4;
    // constexpr int K_BLOCK_SIZE = 2 * KQ_HEAD_BLOCK * (K_BLOCK+1) * sizeof(T_smem);
    constexpr int SOFTMAX_BLOCK_SIZE = K_BLOCK * (Q_BLOCK+4) * sizeof(T_smem);
    // constexpr int SOFTMAX_BLOCK_SIZE = V_BLOCK * (Q_BLOCK+4) * sizeof(T_smem);
    constexpr int LDV = V_BLOCK + 4;
    constexpr int V_BLOCK_SIZE = 2 * VS_HEAD_BLOCK * LDV * sizeof(T_smem);
    constexpr int A_BLOCK_SIZE = VS_HEAD_BLOCK * (Q_BLOCK+4) * sizeof(T_smem);

    // Assign pointers to shared memory regions with proper offsets
    T_smem* max_block = reinterpret_cast<T_smem*>(shared_mem);
    T_smem* sumexp_block = reinterpret_cast<T_smem*>(shared_mem + MAX_BLOCK_SIZE);
    bool* mask_block = reinterpret_cast<bool*>(shared_mem);
    T_smem* Q_block = reinterpret_cast<T_smem*>(shared_mem);
    T_smem* K_block = reinterpret_cast<T_smem*>(shared_mem + Q_BLOCK_SIZE);
    T_smem* softmax_block = reinterpret_cast<T_smem*>(shared_mem);
    T_smem* V_block = reinterpret_cast<T_smem*>(shared_mem + SOFTMAX_BLOCK_SIZE);
    T_smem* A_block = reinterpret_cast<T_smem*>(shared_mem + SOFTMAX_BLOCK_SIZE + V_BLOCK_SIZE);

    // Helper functions for indexing into the 1D arrays
    auto mask_idx = [&](int q, int k) -> int {
        return q * (K_BLOCK+4) + k;
    };

    auto Q_idx = [&](int buf, int h, int q) -> int {
        return buf * KQ_HEAD_BLOCK * LDQ + h * LDQ + q;
    };

    auto K_idx = [&](int buf, int h, int k) -> int {
        return buf * KQ_HEAD_BLOCK * LDK + h * LDK + k;
    };

    auto softmax_idx = [&](int k, int q) -> int {
        return k * (Q_BLOCK+4) + q;
    };

    auto V_idx = [&](int buf, int h, int k) -> int {
        return buf * VS_HEAD_BLOCK * LDV + h * LDV + k;
        // return buf * (VS_HEAD_BLOCK+1) * (V_BLOCK) + h + (VS_HEAD_BLOCK+1) * k;
    };

    auto A_idx = [&](int h, int q) -> int {
        return h * (Q_BLOCK+4) + q;
    };

    // Number of tiles of a block of softmax(mask(K'Q)) in each dimension
    constexpr int KQ_K_TILE_NUM = K_BLOCK / KQ_K_TILE;
    constexpr int KQ_Q_TILE_NUM = Q_BLOCK / KQ_Q_TILE;
    constexpr int KQ_TILE_NUM = KQ_K_TILE_NUM * KQ_Q_TILE_NUM;

    // Number of tiles of a block of VS in each dimension
    constexpr int VS_HEAD_TILE_NUM = VS_HEAD_BLOCK / VS_HEAD_TILE;
    constexpr int VS_Q_TILE_NUM = Q_BLOCK / VS_Q_TILE;
    constexpr int VS_TILE_NUM = VS_HEAD_TILE_NUM * VS_Q_TILE_NUM;

    // Number of tiles of softmax(mask(K'Q)) per warp in a block
    constexpr int KQ_TILE_PER_WARP = (KQ_TILE_NUM + NUM_WARPS - 1) / NUM_WARPS;

    // Number of tiles of V @ softmax per warp in a block
    constexpr int VS_TILE_PER_WARP = (VS_TILE_NUM + NUM_WARPS - 1) / NUM_WARPS;

    // Warp for softmax(mask(K'Q)) tile is the following grid of threads
    constexpr int KQ_WARP_K_THREADS = 8;
    constexpr int KQ_WARP_Q_THREADS = 32 / KQ_WARP_K_THREADS;

    // Warp for V @ softmax tile is the following grid of threads
    constexpr int VS_WARP_HEAD_THREADS = 8;
    constexpr int VS_WARP_Q_THREADS = 32 / VS_WARP_HEAD_THREADS;

    // Number of softmax(mask(K'Q)) tile elements per thread
    constexpr int KQ_TILE_K_PER_THREAD = KQ_K_TILE / KQ_WARP_K_THREADS;
    constexpr int KQ_TILE_Q_PER_THREAD = KQ_Q_TILE / KQ_WARP_Q_THREADS;

    // Number of V @ softmax tile elements per thread
    constexpr int VS_TILE_HEAD_PER_THREAD =
        VS_HEAD_TILE / VS_WARP_HEAD_THREADS;
    constexpr int VS_TILE_Q_PER_THREAD = VS_Q_TILE / VS_WARP_Q_THREADS;

    // Thread-local registers for max and sumexp of softmax(mask(K'Q))
    T_accum max_reg[KQ_TILE_PER_WARP][KQ_TILE_Q_PER_THREAD];
    T_accum sumexp_reg[KQ_TILE_PER_WARP][KQ_TILE_Q_PER_THREAD];

    for (int i = 2 * thread_id; i < Q_BLOCK; i += 2 * block_size)
    {
        float4 maxsumexp_val = *reinterpret_cast<const float4*>(
            &maxsumexp[2 * (i + q_block_idx * Q_BLOCK + seq * batch_idx)]);
        max_block[i] = maxsumexp_val.x;
        sumexp_block[i] = 1.0 / maxsumexp_val.y; // inverse of sumexp
        max_block[i+1] = maxsumexp_val.z;
        sumexp_block[i+1] = 1.0 / maxsumexp_val.w; // inverse of sumexp
    }
    __syncthreads();

    // int magic_number = (warp_id % KQ_Q_TILE_NUM) * KQ_Q_TILE;
    // magic_number += lane_id % KQ_WARP_Q_THREADS;
    //#pragma unroll
    for (int tile_idx_loop = 0; tile_idx_loop < KQ_TILE_PER_WARP;
            ++tile_idx_loop)
    {
        int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
        int q_tile_idx = (tile_idx % KQ_Q_TILE_NUM);
        int thread_q_idx = lane_id % KQ_WARP_Q_THREADS;
        int q_local = q_tile_idx * KQ_Q_TILE + thread_q_idx;
        //#pragma unroll
        for (int j = 0; j < KQ_TILE_Q_PER_THREAD; ++j)
        {
            int q_idx = q_local + j * KQ_WARP_Q_THREADS;
            max_reg[tile_idx_loop][j] = max_block[q_idx];
            sumexp_reg[tile_idx_loop][j] = sumexp_block[q_idx];
            // max_reg[tile_idx_loop][j] = max_block[magic_number + j * KQ_WARP_Q_THREADS];
            // sumexp_reg[tile_idx_loop][j] = sumexp_block[magic_number + j * KQ_WARP_Q_THREADS];
        }
    }
    __syncthreads();

    // Process K,V blocks
    for (Index kv_block_idx = kv_block_start; kv_block_idx < kv_block_end;
            kv_block_idx += K_BLOCK)
    {
        T_accum softmax_reg[KQ_TILE_PER_WARP][KQ_TILE_K_PER_THREAD][
            KQ_TILE_Q_PER_THREAD];
        // Stage 1: Compute softmax(mask(K'Q))
        {
            // Thread-local registers for softmax(mask(K'Q))
            // T_accum softmax_reg[KQ_TILE_PER_WARP][KQ_TILE_K_PER_THREAD][
            //     KQ_TILE_Q_PER_THREAD];

            // Initialize buffer index for double buffering
            int buf_idx = 0;

            // Initialize mask tile
            int j = thread_id % Q_BLOCK;
            //#pragma unroll
            for (int i = 16 * (thread_id / Q_BLOCK); i < K_BLOCK;
                    i += 16 * (block_size / Q_BLOCK))
            {
                float4 mask_val = *reinterpret_cast<const float4*>(
                    &mask[kv_block_idx + i + (j + q_block_idx * Q_BLOCK) * seq]);
                bool *mask_val_bool = reinterpret_cast<bool*>(&mask_val);
                for (int k = 0; k < 16; ++k)
                {
                    mask_block[mask_idx(j, i+k)] = bool(mask_val_bool[k]);
                }
            }
            __syncthreads();

            // Initialize K'Q block on registers with mask information
            // We do it the same way as gemm K'Q to ensure maximal register usage
            //#pragma unroll
            for(int tile_idx_loop = 0; tile_idx_loop < KQ_TILE_PER_WARP;
                    ++tile_idx_loop)
            {
                int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
                int q_tile_idx = (tile_idx % KQ_Q_TILE_NUM);
                int k_tile_idx = (tile_idx / KQ_Q_TILE_NUM);
                int thread_q_idx = lane_id % KQ_WARP_Q_THREADS;
                int thread_k_idx = lane_id / KQ_WARP_Q_THREADS;
                int q_local = q_tile_idx * KQ_Q_TILE + thread_q_idx;
                int k_local = k_tile_idx * KQ_K_TILE + thread_k_idx;
                //#pragma unroll
                for (int i = 0; i < KQ_TILE_K_PER_THREAD; ++i)
                {
                    //#pragma unroll
                    for (int j = 0; j < KQ_TILE_Q_PER_THREAD; ++j)
                    {
                        if (mask_block[mask_idx(q_local + KQ_WARP_Q_THREADS * j,
                                k_local + KQ_WARP_K_THREADS * i)])
                        // if (mask_block[mask_idx(magic_number + KQ_WARP_Q_THREADS * j,
                        //         k_local + KQ_WARP_K_THREADS * i)])
                        {
                            softmax_reg[tile_idx_loop][i][j] = 0;
                        }
                        else
                        {
                            softmax_reg[tile_idx_loop][i][j] =
                                -std::numeric_limits<T_accum>::infinity();
                        }
                    }
                }
            }

            __syncthreads();

            // Load the first Q block of shape KQ_HEAD_BLOCK x Q_BLOCK
            gmem_to_smem_transposed_vec4<T_gmem, T_smem, KQ_HEAD_BLOCK, Q_BLOCK>(
                Q + HEAD_SIZE * (q_block_idx * Q_BLOCK + seq * batch_idx),
                Q_block + Q_idx(buf_idx, 0, 0),
                HEAD_SIZE,
                LDQ,
                thread_id,
                block_size
            );

            // Load the first K block of shape KQ_HEAD_BLOCK x K_BLOCK
            gmem_to_smem_transposed_vec4<T_gmem, T_smem, KQ_HEAD_BLOCK, K_BLOCK>(
                K + HEAD_SIZE * (kv_block_idx + seq * batch_idx),
                K_block + K_idx(buf_idx, 0, 0),
                HEAD_SIZE,
                LDK,
                thread_id,
                block_size
            );

            // Wait for all threads to load the first K and Q blocks
            __syncthreads();

            // Process head dimension in chunks to compute entire block of K'Q
            //#pragma unroll 1
            for (int head_offset = 0; head_offset < HEAD_SIZE;
                    head_offset += KQ_HEAD_BLOCK)
            {
                // Buffer index for next iteration
                int next_buf_idx = 1 - buf_idx;

                // Load next Q and K blocks
                if (head_offset + KQ_HEAD_BLOCK < HEAD_SIZE)
                {
                    // Load next Q block of shape KQ_HEAD_BLOCK x Q_BLOCK
                    gmem_to_smem_transposed_vec4<
                        T_gmem, T_smem, KQ_HEAD_BLOCK, Q_BLOCK>(
                        Q + HEAD_SIZE * (q_block_idx * Q_BLOCK + seq * batch_idx)
                            + (head_offset + KQ_HEAD_BLOCK),
                        Q_block + Q_idx(next_buf_idx, 0, 0),
                        HEAD_SIZE,
                        LDQ,
                        thread_id,
                        block_size
                    );

                    // Load next K block of shape KQ_HEAD_BLOCK x K_BLOCK
                    gmem_to_smem_transposed_vec4<
                        T_gmem, T_smem, KQ_HEAD_BLOCK, K_BLOCK>(
                        K + HEAD_SIZE * (kv_block_idx + seq * batch_idx)
                            + (head_offset + KQ_HEAD_BLOCK),
                        K_block + K_idx(next_buf_idx, 0, 0),
                        HEAD_SIZE,
                        LDK,
                        thread_id,
                        block_size
                    );
                }

                // Accumulate block of K'Q
                //#pragma unroll
                for (int tile_idx_loop = 0; tile_idx_loop < KQ_TILE_PER_WARP;
                        ++tile_idx_loop)
                {
                    int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
                    int q_tile_idx = (tile_idx % KQ_Q_TILE_NUM);
                    int k_tile_idx = (tile_idx / KQ_Q_TILE_NUM);
                    int thread_q_idx = lane_id % KQ_WARP_Q_THREADS;
                    int thread_k_idx = lane_id / KQ_WARP_Q_THREADS;
                    int q = q_tile_idx * KQ_Q_TILE + thread_q_idx;
                    int k = k_tile_idx * KQ_K_TILE + thread_k_idx;
                    #pragma unroll 8
                    for (int h = 0; h < KQ_HEAD_BLOCK; ++h)
                    {
                        float a_vals[KQ_TILE_K_PER_THREAD],
                            b_vals[KQ_TILE_Q_PER_THREAD];
                        #pragma unroll
                        for (int i = 0; i < KQ_TILE_K_PER_THREAD; ++i)
                        {
                            // Load from K_block (it is transposed)
                            a_vals[i] = K_block[K_idx(buf_idx, h,
                                k + KQ_WARP_K_THREADS * i)];
                        }
                        #pragma unroll
                        for (int j = 0; j < KQ_TILE_Q_PER_THREAD; ++j)
                        {
                            // Load from Q_block
                            b_vals[j] = Q_block[Q_idx(buf_idx, h,
                                q + KQ_WARP_Q_THREADS * j)];
                        }
                        #pragma unroll
                        for (int i = 0; i < KQ_TILE_K_PER_THREAD; ++i)
                        {
                            #pragma unroll
                            for (int j = 0; j < KQ_TILE_Q_PER_THREAD; ++j)
                            {
                                softmax_reg[tile_idx_loop][i][j] +=
                                    a_vals[i] * b_vals[j];
                            }
                        }
                    }
                }

                __syncthreads();

                // Swap buffers for next iteration
                buf_idx = 1 - buf_idx;
            }

            // Apply softmax to thread-local registers and write results to shared memory
            //#pragma unroll
            for (int tile_idx_loop = 0; tile_idx_loop < KQ_TILE_PER_WARP;
                    ++tile_idx_loop)
            {
                int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
                int q_tile_idx = (tile_idx % KQ_Q_TILE_NUM);
                int k_tile_idx = (tile_idx / KQ_Q_TILE_NUM);
                int thread_q_idx = lane_id % KQ_WARP_Q_THREADS;
                int thread_k_idx = lane_id / KQ_WARP_Q_THREADS;
                int q = q_tile_idx * KQ_Q_TILE + thread_q_idx;
                int k = k_tile_idx * KQ_K_TILE + thread_k_idx;
                #pragma unroll
                for (int j = 0; j < KQ_TILE_Q_PER_THREAD; ++j)
                {
                    const T_accum max_val = max_reg[tile_idx_loop][j];
                    const T_accum sumexp = sumexp_reg[tile_idx_loop][j];
                    #pragma unroll
                    for (int i = 0; i < KQ_TILE_K_PER_THREAD; ++i)
                    {
                        T_accum val = scale * softmax_reg[tile_idx_loop][i][j];
                        softmax_reg[tile_idx_loop][i][j] = sumexp
                            * ::exp(val - max_val);
                        softmax_block[softmax_idx(k + KQ_WARP_K_THREADS * i,
                            // magic_number + KQ_WARP_Q_THREADS * j)] =
                            q + KQ_WARP_Q_THREADS * j)] =
                            T_smem{softmax_reg[tile_idx_loop][i][j]};
                    }
                }
            }
        } // End of stage 1

        // Stage 2: Compute VS
        {
            // Thread-local registers for the output VS
            // T_accum A_reg[VS_TILE_PER_WARP][VS_TILE_HEAD_PER_THREAD][
            //     VS_TILE_Q_PER_THREAD];

            // Since VS is of shape HEAD_SIZE x Q_BLOCK, we process head
            // dimension in chunks of VS_HEAD_BLOCK. These chunks are
            // independent, they are processed sequentially one by one.
            //#pragma unroll 2
            for (int head_offset = 0; head_offset < HEAD_SIZE;
                    head_offset += VS_HEAD_BLOCK)
            {
                int buf_idx = 0;

                // Load the first V block of shape VS_HEAD_BLOCK x V_BLOCK
                gmem_to_smem_transposed_vec4<T_gmem, T_smem, VS_HEAD_BLOCK, V_BLOCK>(
                    V + HEAD_SIZE * (kv_block_idx + seq * batch_idx)
                        + head_offset,
                    V_block + V_idx(buf_idx, 0, 0),
                    HEAD_SIZE,
                    LDV,
                    thread_id,
                    block_size);

                // Clear the output registers
                // #pragma unroll
                for (int tile_idx_loop = 0; tile_idx_loop < VS_TILE_PER_WARP;
                        ++tile_idx_loop)
                {
                    #pragma unroll
                    for (int i = 0; i < VS_TILE_HEAD_PER_THREAD; ++i)
                    {
                        #pragma unroll
                        for (int j = 0; j < VS_TILE_Q_PER_THREAD; ++j)
                        {
                            // A_reg[tile_idx_loop][i][j] = 0.0;
                            softmax_reg[tile_idx_loop][i][j] = 0.0;
                        }
                    }
                }

                __syncthreads();

                // Here we process multiplication of V block of shape
                // VS_HEAD_BLOCK x K_BLOCK by S block of shape
                // K_BLOCK x Q_BLOCK. We do it in chunks of size V_BLOCK
                // along dimension of size K_BLOCK.
                // #pragma unroll 2
                for (int v_block_idx = 0; v_block_idx < K_BLOCK;
                        v_block_idx += V_BLOCK)
                {
                    // Prefetch the next V block if not at the last iteration
                    if (v_block_idx + V_BLOCK < K_BLOCK)
                    {
                        int next_buf_idx = 1 - buf_idx;
                        int next_v_idx_start = v_block_idx + V_BLOCK
                            + kv_block_idx + seq * batch_idx;
                        gmem_to_smem_transposed_vec4<
                            T_gmem, T_smem, VS_HEAD_BLOCK, V_BLOCK>(
                            V + HEAD_SIZE * next_v_idx_start + head_offset,
                            V_block + V_idx(next_buf_idx, 0, 0),
                            HEAD_SIZE,
                            LDV,
                            thread_id,
                            block_size);
                    }

                    // Process tiles in a round-robin fashion across warps
                    // #pragma unroll
                    for (int tile_idx_loop = 0; tile_idx_loop < VS_TILE_PER_WARP;
                            ++tile_idx_loop)
                    {
                        int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
                        int head_tile_idx = (tile_idx % VS_HEAD_TILE_NUM);
                        int q_tile_idx = (tile_idx / VS_HEAD_TILE_NUM);
                        // int thread_head_idx = lane_id % VS_WARP_HEAD_THREADS;
                        // int thread_q_idx = lane_id / VS_WARP_HEAD_THREADS;
                        int thread_head_idx = lane_id / VS_WARP_Q_THREADS;
                        int thread_q_idx = lane_id % VS_WARP_Q_THREADS;
                        int h = head_tile_idx * VS_HEAD_TILE + thread_head_idx;
                        int q = q_tile_idx * VS_Q_TILE + thread_q_idx;
                        #pragma unroll 8
                        for (int v = 0; v < V_BLOCK; ++v)
                        {
                            float a_vals[VS_TILE_HEAD_PER_THREAD],
                                b_vals[VS_TILE_Q_PER_THREAD];
                            #pragma unroll
                            for (int i = 0; i < VS_TILE_HEAD_PER_THREAD; ++i)
                            {
                                // Load from V_block
                                a_vals[i] = V_block[V_idx(buf_idx,
                                        h + VS_WARP_HEAD_THREADS * i, v)];
                            }
                            #pragma unroll
                            for (int j = 0; j < VS_TILE_Q_PER_THREAD; ++j)
                            {
                                // Load from softmax_block
                                b_vals[j] = softmax_block[softmax_idx(
                                        v + v_block_idx,
                                        q + VS_WARP_Q_THREADS * j)];
                            }
                            #pragma unroll
                            for (int i = 0; i < VS_TILE_HEAD_PER_THREAD; ++i)
                            {
                                #pragma unroll
                                for (int j = 0; j < VS_TILE_Q_PER_THREAD; ++j)
                                {
                                    // A_reg[tile_idx_loop][i][j] +=
                                    softmax_reg[tile_idx_loop][i][j] +=
                                        T_accum{a_vals[i] * b_vals[j]};
                                }
                            }
                        }
                    }

                    // Wait for all threads in the warp to finish before processing the next block
                    __syncthreads();

                    // Swap buffers for next iteration
                    buf_idx = 1 - buf_idx;
                }

                // Process tiles in a round-robin fashion across warps
                // #pragma unroll
                for (int tile_idx_loop = 0; tile_idx_loop < VS_TILE_PER_WARP;
                        ++tile_idx_loop)
                {
                    int tile_idx = warp_id + tile_idx_loop * NUM_WARPS;
                    int head_tile_idx = (tile_idx % VS_HEAD_TILE_NUM);
                    int q_tile_idx = (tile_idx / VS_HEAD_TILE_NUM);
                    int thread_head_idx = lane_id % VS_WARP_HEAD_THREADS;
                    int thread_q_idx = lane_id / VS_WARP_HEAD_THREADS;
                    int h = head_tile_idx * VS_HEAD_TILE + thread_head_idx;
                    int q = q_tile_idx * VS_Q_TILE + thread_q_idx;
                    #pragma unroll
                    for (int i = 0; i < VS_TILE_HEAD_PER_THREAD; ++i)
                    {
                        #pragma unroll
                        for (int j = 0; j < VS_TILE_Q_PER_THREAD; ++j)
                        {
                            const Index head_idx_local = h + VS_WARP_HEAD_THREADS * i;
                            const Index q_idx_local = q + VS_WARP_Q_THREADS * j;
                            const Index head_idx = head_offset + head_idx_local;
                            const Index q_idx = q_block_idx * Q_BLOCK + q_idx_local;
                            const Index a_idx = head_idx + HEAD_SIZE * (q_idx + seq * batch_idx);
                            // atomicAdd(&A[a_idx], T_gmem{A_reg[tile_idx_loop][i][j]});
                            atomicAdd(&A[a_idx], T_gmem{softmax_reg[tile_idx_loop][i][j]});
                            // A_block[A_idx(head_idx_local, q_idx_local)] =
                            //      T_smem{A_reg[tile_idx_loop][i][j]};
                        }
                    }
                }
                __syncthreads();
            }
        } // End of stage 2
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template<typename T>
void cuda(hipStream_t stream, Index batch, Index seq, Index head,
          const T *K, const T *Q, const bool_t *mask, const T *maxsumexp,
          const T *V, T *A) noexcept
{
    // Define block and grid sizes
    constexpr int NUM_THREADS = 128;  // Total number of threads per block
    constexpr int NUM_WARPS = NUM_THREADS / 32; // Number of warps per block

    // Ensure we have the right number of threads for the warps
    static_assert(NUM_THREADS % 32 == 0, "NUM_THREADS must be a multiple of 32 (warp size)");

    // K'Q matmul is done by blocks:
    // K is split into blocks of size KQ_HEAD_BLOCK x K_BLOCK
    // Q is split into blocks of size KQ_HEAD_BLOCK x Q_BLOCK
    // K'Q is split into blocks of size K_BLOCK x Q_BLOCK
    constexpr int Q_BLOCK = 64;
    constexpr int K_BLOCK = 128;
    constexpr int KQ_HEAD_BLOCK = 16;

    // V @ softmax is done by blocks:
    // V is split into blocks of size VS_HEAD_BLOCK x V_BLOCK
    // softmax is split into blocks of size V_BLOCK x Q_BLOCK
    // V @ softmax is split into blocks of size VS_HEAD_BLOCK x Q_BLOCK
    constexpr int V_BLOCK = 16;
    constexpr int VS_HEAD_BLOCK = 64;

    // Split K and V into KV_SPLIT parts, each part is processed by a different
    // CUDA block. This is done to balance between parallelism and overhead.
    constexpr int KV_SPLIT = 4;

    // Calculate shared memory size
    constexpr int LDQ = Q_BLOCK + 4;
    constexpr int LDK = K_BLOCK + 4;
    constexpr int LDV = V_BLOCK + 4;
    constexpr int Q_BLOCK_SIZE = 2 * KQ_HEAD_BLOCK * LDQ * sizeof(float);
    constexpr int K_BLOCK_SIZE = 2 * KQ_HEAD_BLOCK * LDK * sizeof(float);
    constexpr int SOFTMAX_BLOCK_SIZE = K_BLOCK * (Q_BLOCK+4) * sizeof(float);
    constexpr int V_BLOCK_SIZE = 2 * VS_HEAD_BLOCK * LDV * sizeof(float);
    constexpr int A_BLOCK_SIZE = VS_HEAD_BLOCK * (Q_BLOCK+4) * sizeof(float);
    constexpr int SHARED_MEM_SIZE = std::max(Q_BLOCK_SIZE + K_BLOCK_SIZE,
            SOFTMAX_BLOCK_SIZE + V_BLOCK_SIZE);

    constexpr int KQ_Q_TILE = 32;
    constexpr int KQ_K_TILE = 64;
    constexpr int VS_HEAD_TILE = 32;
    constexpr int VS_Q_TILE = 32;
    static_assert(K_BLOCK * Q_BLOCK >= KQ_Q_TILE * KQ_K_TILE * NUM_WARPS,
            "K_BLOCK * Q_BLOCK must be greater than KQ_Q_TILE * KQ_K_TILE "
            "* NUM_WARPS");
    static_assert(Q_BLOCK * VS_HEAD_BLOCK >= VS_Q_TILE * VS_HEAD_TILE * NUM_WARPS,
            "Q_BLOCK * VS_HEAD_BLOCK must be greater than VS_Q_TILE * VS_HEAD_TILE "
            "* NUM_WARPS");

    // Use 1D thread blocks instead of 2D
    dim3 threads(NUM_THREADS);
    dim3 blocks((seq + Q_BLOCK - 1) / Q_BLOCK, batch, KV_SPLIT);

    // Calculate scaling factor
    using Y = typename T::repr_t;
    T scale = T(Y(1.0) / std::sqrt(Y(head)));

    // Clear the output
    hipMemsetAsync(A, 0, batch * head * seq * sizeof(T), stream);

    // Launch kernel based on head size
    if (head == 64)
    {
        constexpr int HEAD_SIZE = 64;

        if constexpr (std::is_same_v<T, nntile::fp32_t>)
        {
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                flash_softmax_gemm_kernel<float), float, float,
                    HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
                    KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
                    VS_Q_TILE, KV_SPLIT, NUM_WARPS>,
                    hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM_SIZE);

            flash_softmax_gemm_kernel<float, float, float,
                    HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
                    KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
                    VS_Q_TILE, KV_SPLIT, NUM_WARPS>
                <<<blocks, threads, SHARED_MEM_SIZE, stream>>>(batch, seq, scale.value,
                    reinterpret_cast<const float*>(K), reinterpret_cast<const float*>(Q), mask,
                    reinterpret_cast<const float*>(maxsumexp), reinterpret_cast<const float*>(V),
                    reinterpret_cast<float*>(A));
            gpuErrchk( hipPeekAtLastError() );
        }
        else
        {
            std::cerr << "Unsupported type: " << typeid(T).name() << std::endl;
        }
        // TODO: enable other types T later
    } // TODO: enable other heads later
    else if (head == 128)
    {
        constexpr int HEAD_SIZE = 128;
        if constexpr (std::is_same_v<T, nntile::fp32_t>)
        {
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                flash_softmax_gemm_kernel<float), float, float,
                    HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
                    KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
                    VS_Q_TILE, KV_SPLIT, NUM_WARPS>,
                    hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM_SIZE);

            flash_softmax_gemm_kernel<float, float, float,
                    HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
                    KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
                    VS_Q_TILE, KV_SPLIT, NUM_WARPS>
                <<<blocks, threads, SHARED_MEM_SIZE, stream>>>(batch, seq, scale.value,
                    reinterpret_cast<const float*>(K), reinterpret_cast<const float*>(Q), mask,
                    reinterpret_cast<const float*>(maxsumexp), reinterpret_cast<const float*>(V),
                    reinterpret_cast<float*>(A));
            gpuErrchk( hipPeekAtLastError() );
        }
        else
        {
            std::cerr << "Unsupported type: " << typeid(T).name() << std::endl;
        }
        // TODO: enable other types T later
    }
    else if (head == 256)
    {
        constexpr int HEAD_SIZE = 256;
        if constexpr (std::is_same_v<T, nntile::fp32_t>)
        {
            // hipFuncSetAttribute(reinterpret_cast<const void*>(
            //     flash_softmax_gemm_kernel<float), float, float,
            //         HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
            //         KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
            //         VS_Q_TILE, KV_SPLIT, NUM_WARPS>,
            //         hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM_SIZE);

            flash_softmax_gemm_kernel<float, float, float,
                    HEAD_SIZE, Q_BLOCK, K_BLOCK, KQ_HEAD_BLOCK, KQ_Q_TILE,
                    KQ_K_TILE, VS_HEAD_BLOCK, V_BLOCK, VS_HEAD_TILE,
                    VS_Q_TILE, KV_SPLIT, NUM_WARPS>
                <<<blocks, threads, SHARED_MEM_SIZE, stream>>>(batch, seq, scale.value,
                    reinterpret_cast<const float*>(K), reinterpret_cast<const float*>(Q), mask,
                    reinterpret_cast<const float*>(maxsumexp), reinterpret_cast<const float*>(V),
                    reinterpret_cast<float*>(A));
            gpuErrchk( hipPeekAtLastError() );
        }
        else
        {
            std::cerr << "Unsupported type: " << typeid(T).name() << std::endl;
        }
        // TODO: enable other types T later
    }
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const fp32_t *K, const fp32_t *Q, const bool_t *mask,
        const fp32_t *maxsumexp, const fp32_t *V, fp32_t *A) noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const fp64_t *K, const fp64_t *Q, const bool_t *mask,
        const fp64_t *maxsumexp, const fp64_t *V, fp64_t *A) noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const bf16_t *K, const bf16_t *Q, const bool_t *mask,
        const bf16_t *maxsumexp, const bf16_t *V, bf16_t *A) noexcept;

template
void cuda<fp32_fast_tf32_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const fp32_fast_tf32_t *K, const fp32_fast_tf32_t *Q, const bool_t *mask,
        const fp32_fast_tf32_t *maxsumexp, const fp32_fast_tf32_t *V,
        fp32_fast_tf32_t *A) noexcept;

template
void cuda<fp32_fast_fp16_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const fp32_fast_fp16_t *K, const fp32_fast_fp16_t *Q, const bool_t *mask,
        const fp32_fast_fp16_t *maxsumexp, const fp32_fast_fp16_t *V,
        fp32_fast_fp16_t *A) noexcept;

template
void cuda<fp32_fast_bf16_t>(hipStream_t stream, Index batch, Index seq, Index head,
        const fp32_fast_bf16_t *K, const fp32_fast_bf16_t *Q, const bool_t *mask,
        const fp32_fast_bf16_t *maxsumexp, const fp32_fast_bf16_t *V,
        fp32_fast_bf16_t *A) noexcept;

} // namespace nntile::kernel::flash_softmax_gemm
