#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/gelu/cuda.cu
 * GeLU operation on a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/gelu/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::gelu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::repr_t;
    constexpr Y mone = -1, pt5 = 0.5;
    const Y f1 = mone / sqrt(Y{2.0});

    if(i < nelems)
    {
        Y z = Y{data[i]};
        Y y = erfc(f1 * z);
        data[i] = T{pt5 * z * y};
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data)
    noexcept
//! Inplace GeLU operation performed on CUDA
/*! Uses very slow std::erfc() function, so consider using approximated version
 * nntile::kernel::cpu::gelutanh(). Does the following per-element operation:
 * GeLU(z) = 0.5 z erfc(-z/sqrt(2))
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data: Buffer to apply GeLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

    template
void cuda<bf16_t>(hipStream_t stream, Index nelems, bf16_t *data)
    noexcept;

} // namespace nntile::kernel::gelu
