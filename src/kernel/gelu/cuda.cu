#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/gelu/cuda.cu
 * GeLU operation on a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include "nntile/kernel/gelu/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::gelu
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, T *data)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    constexpr T mone = -1, pt5 = 0.5;
    const T f1 = mone / sqrt(T{2.0});
    if(i < nelems)
    {
        T z = data[i];
        T y = erfc(f1 * z);
        data[i] = pt5 * z * y;
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, T *data_)
    noexcept
//! Inplace GeLU operation performed on CUDA
/*! Uses very slow std::erfc() function, so consider using approximated version
 * nntile::kernel::cpu::gelutanh(). Does the following per-element operation:
 * GeLU(z) = 0.5 z erfc(-z/sqrt(2))
 *
 * @params[in] nelems: Number of elements in a buffer
 * @params[inout] data_: Buffer to apply GeLU
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    using Y = typename CUDAComputeType<T>::value;
    auto data = reinterpret_cast<Y *>(data_);
    (cuda_kernel<Y>)<<<blocks, threads, 0, stream>>>(nelems, data);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, fp32_t *data)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, fp64_t *data)
    noexcept;

} // namespace nntile::kernel::gelu
