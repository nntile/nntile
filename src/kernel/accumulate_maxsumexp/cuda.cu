#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/accumulate_maxsumexp/cuda.cu
 * Accumulate maxsumexp buffers on CUDA
 *
 * @version 1.0.0
 * */

#include "nntile/kernel/accumulate_maxsumexp/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::accumulate_maxsumexp
{

template<typename T>
static __global__
void cuda_kernel(Index nelems, const T *src_, T *dst_)
//! Accumulate two maxsumexp buffers on CUDA
/*! Performs the following operation:
 *      dst[2*i+1] = dst[2*i+1]*exp(dst[2*i]) + src[2*i+1]*exp(src[2*i]),
 *      dst[2*i] = max(src[2*i], dst[2*i]).
 *
 * @param[in] nelems: Number of (max,sumexp) pairs of the src and dst tensors
 * @param[in] src: Source tensor
 * @param[inout] dst: Destination of the maxsumexp accumulation
 * */
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    using Y = typename T::compat_t;
    using Z = typename CUDAComputeType<T>::value;
    auto src = reinterpret_cast<const Z *>(src_);
    auto dst = reinterpret_cast<Z *>(dst_);
    constexpr Y zero = 0.0;
    Y dst_odd = Y{dst[2*i+1]};
    Y dst_even = Y{dst[2*i]};

    Y src_odd = Y{src[2*i+1]};
    Y src_even = Y{src[2*i]};
    if(i < nelems)
    {
        // Do nothing if sum of exponents of source is zero
        if(src_odd != zero)
        {
            // Overwrite if old value of sum is zero
            if(dst_odd == zero)
            {
                dst[2*i] = src[2*i];
                dst[2*i+1] = src[2*i+1];
            }
            // Otherwise update based on maximum
            else if(dst_even < src_even)
            {
                dst[2*i+1] = Z{src_odd + dst_odd * ::exp(dst_even - src_even)};
                dst[2*i] = src[2*i];
            }
            else
            {
                dst[2*i+1] = Z{dst_odd + src_odd * ::exp(src_even - dst_even)};            
            }
        }
    }
}

template<typename T>
void cuda(hipStream_t stream, Index nelems, const T *src_, T *dst_)
    noexcept
//! Accumulate two maxsumexp buffers on CUDA
/*! Performs the following operation:
 *      dst[2*i+1] = dst[2*i+1]*exp(dst[2*i]) + src[2*i+1]*exp(src[2*i]),
 *      dst[2*i] = max(src[2*i], dst[2*i]).
 *
 * @param[in] nelems: Number of (max,sumexp) pairs of the src and dst tensors
 * @param[in] src_: Source tensor
 * @param[inout] dst_: Destination of the maxsumexp accumulation
 * */
{
    dim3 blocks((nelems+255)/256), threads(256);
    // auto src = cast_pointer_cuda<T>(src_);
    // auto dst = cast_pointer_cuda<T>(dst_);
    // using Y = typename CUDAComputeType<T>::value;
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(nelems, src_, dst_);
}

// Explicit instantiation
template
void cuda<fp32_t>(hipStream_t stream, Index nelems, const fp32_t *src,
        fp32_t *dst)
    noexcept;

template
void cuda<fp64_t>(hipStream_t stream, Index nelems, const fp64_t *src,
        fp64_t *dst)
    noexcept;

template
void cuda<bf16_t>(hipStream_t stream, Index nelems, const bf16_t *src,
        bf16_t *dst)
    noexcept;

} // namespace nntile::kernel::accumulate_maxsumexp
