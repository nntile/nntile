#include "hip/hip_runtime.h"
/*! @copyright (c) 2022-present Skolkovo Institute of Science and Technology
 *                              (Skoltech), Russia. All rights reserved.
 *                 2023-present Artificial Intelligence Research Institute
 *                              (AIRI), Russia. All rights reserved.
 *
 * NNTile is software framework for fast training of big neural networks on
 * distributed-memory heterogeneous systems based on StarPU runtime system.
 *
 * @file src/kernel/maxsumexp/cuda.cu
 * Max and sum of exponents of a buffer on CUDA
 *
 * @version 1.1.0
 * */

#include <iostream>

#include "nntile/kernel/maxsumexp/cuda.hh"
#include "nntile/kernel/cuda.hh"

namespace nntile::kernel::maxsumexp
{

/**
 * This implementation is taken from 3c4a8ee08f66732d67789f851c6bff788e41fd38.
 */
// clang-format off
template<typename T>
static __global__
void cuda_kernel(Index m, Index m_per_block, Index n, Index n_per_block,
        Index k, Index mk, const T * __restrict__ src,
        T * __restrict__ maxsumexp)
{
    Index i1_block = blockIdx.y, i2_block = blockIdx.z,
          i0_start = threadIdx.x, i0_step = blockDim.x;

    using Y = typename T::repr_t;
    constexpr Y zero = 0.0, one = 1.0;
    if(i0_start < k)
    {
        for(Index i1 = i1_block*m_per_block;
                i1 < (i1_block+1)*m_per_block and i1 < m; ++i1)
        {
            for(Index i2 = i2_block*n_per_block;
                    i2 < (i2_block+1)*n_per_block and i2 < n; ++i2)
            {
                // Get max and sum of exponents of a corresponding slice
                const T *src_slice = src + i2*mk + i1;
                // Init max and sum
                Y max_val = Y{src_slice[i0_start*m]};
                Y sum_val = one;
                // Cycle over slice of input buffer
                for(Index i0 = i0_start+i0_step; i0 < k; i0 += i0_step)
                {
                    // Read value from source
                    Y val = Y{src_slice[i0*m]};
                    // Ignore -inf value, which comes from mask
                    if(::isinf(val))
                    {
                        continue;
                    }
                    // Update max and sum of exponents
                    if(max_val < val)
                    {
                        sum_val = sum_val*(::exp(max_val-val)) + one;
                        max_val = val;
                    }
                    else
                    {
                        sum_val += ::exp(val-max_val);
                    }
                }
                // Per-block of threads max and sum of exponents
                volatile __shared__ Y block_max_val;
                __shared__ Y block_sum_val;
                // Init shared values in the i0_start==0 thread
                if(i0_start == 0)
                {
                    block_max_val = max_val;
                    block_sum_val = zero;
                }
                // Other threads wait until initialization is done
                __syncthreads();
                // Update max at first
                while(block_max_val < max_val)
                {
                    block_max_val = max_val;
                }
                // Sync with all other threads to get per-block max finally
                __syncthreads();
                // Accumulate per-block sum of finite values
                if(not ::isinf(max_val))
                {
                    sum_val *= ::exp(max_val - block_max_val);
                    atomicAdd(&block_sum_val, sum_val);
                }
                __syncthreads();
                // Update output iff per-block sum is not zero
                if(i0_start == 0 and block_sum_val > 0)
                {
                    // Get per-block max and sum of exponents into local variables
                    max_val = block_max_val;
                    sum_val = block_sum_val;
                    Index dst_offset = i1 + i2*m;
                    // Now max_val is finite, we need to accumulate sum of exponents
                    // with the data in global memory
                    Y max_output;
                    Y sum_output = Y{maxsumexp[2*dst_offset+1]};
                    // If data was not yet initialised, just overwrite it
                    if(sum_output == zero)
                    {
                        max_output = max_val;
                        sum_output = sum_val;
                    }
                    // Accumulate otherwise
                    else
                    {
                        max_output = Y{maxsumexp[2*dst_offset]};
                        if(max_val < max_output)
                        {
                            sum_val *= ::exp(max_val - max_output);
                        }
                        else
                        {
                            sum_output *= ::exp(max_output - max_val);
                            max_output = max_val;
                        }
                        sum_output += sum_val;
                    }
                    maxsumexp[2*dst_offset] = T{max_output};
                    maxsumexp[2*dst_offset+1] = T{sum_output};
                }
            }
        }
    }
}

template <typename T>
void LaunchMaxSumExp1(hipStream_t stream, Index m, Index n, Index k,
                      const T *src_, T *maxsumexp_)
    noexcept
{
    // Both source and destination are Fortran-contiguous
    dim3 threads(32, 1, 1), blocks(1, m, n);
    Index m_per_block = 1, n_per_block = 1;
    if(m > 65535)
    {
        m_per_block = (m+65534) / 65535;
        blocks.y = (m+m_per_block-1) / m_per_block;
    }
    if(n > 65535)
    {
        n_per_block = (n+65534) / 65535;
        blocks.z = (n+n_per_block-1) / n_per_block;
    }
    // using Y = typename CUDAComputeType<T>::value;
    // auto src = reinterpret_cast<const Y *>(src_);
    // auto maxsumexp = reinterpret_cast<Y *>(maxsumexp_);
    (cuda_kernel<T>)<<<blocks, threads, 0, stream>>>(m, m_per_block, n,
            n_per_block, k, m*k, src_, maxsumexp_);
}
// clang-format on

template void LaunchMaxSumExp1<fp32_t>(hipStream_t stream, Index m, Index n,
                                       Index k, const fp32_t *src,
                                       fp32_t *maxsumexp) noexcept;

template void LaunchMaxSumExp1<fp64_t>(hipStream_t stream, Index m, Index n,
                                       Index k, const fp64_t *src,
                                       fp64_t *maxsumexp) noexcept;

template void LaunchMaxSumExp1<bf16_t>(hipStream_t stream, Index m, Index n,
                                       Index k, const bf16_t *src,
                                       bf16_t *maxsumexp) noexcept;

//extern __shared__ float extent[]; // User-managed cache on device.
//
//size_t constexpr kMaxBlockSize = 512;
//
//template <typename T, uint32_t kBlockSize>
//__device__ void BlockMaxReduce(volatile T *acc, uint32_t tid) {
//    if constexpr (kBlockSize >= 1024) {
//        if (tid < 512) {
//            acc[tid] = ::fmax(acc[tid], acc[tid + 512]);
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 512) {
//        if (tid < 256) {
//            acc[tid] = ::fmax(acc[tid], acc[tid + 256]);
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 256) {
//        if (tid < 128) {
//            acc[tid] = ::fmax(acc[tid], acc[tid + 128]);
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 128) {
//        if (tid < 64) {
//            acc[tid] = ::fmax(acc[tid], acc[tid + 64]);
//        }
//        __syncthreads();
//    }
//}
//
//template <typename T, uint32_t kBlockSize, uint32_t kStride>
//__device__ void WarpMaxReduceRound(volatile T *acc, uint32_t tid) {
//    if constexpr (kBlockSize >= 2 * kStride) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + kStride]);
//    }
//}
//
//template <typename T, uint32_t kBlockSize>
//__device__ void WarpMaxReduce(volatile T *acc, uint32_t tid) {
//    if constexpr (kBlockSize >= 64) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 32]);
//    }
//    if constexpr (kBlockSize >= 32) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 16]);
//    }
//    if constexpr (kBlockSize >= 16) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 8]);
//    }
//    if constexpr (kBlockSize >= 8) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 4]);
//    }
//    if constexpr (kBlockSize >= 4) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 2]);
//    }
//    if constexpr (kBlockSize >= 2) {
//        acc[tid] = ::fmax(acc[tid], acc[tid + 1]);
//    }
//}
//
//template <typename T, uint32_t kBlockSize>
//__device__ void BlockSumExpReduce(volatile T *acc, uint32_t tid) {
//    if constexpr (kBlockSize >= 1024) {
//        if (tid < 512) {
//            acc[tid] = acc[tid] + acc[tid + 512];
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 512) {
//        if (tid < 256) {
//            acc[tid] = acc[tid] + acc[tid + 256];
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 256) {
//        if (tid < 128) {
//            acc[tid] = acc[tid] + acc[tid + 128];
//        }
//        __syncthreads();
//    }
//    if constexpr (kBlockSize >= 128) {
//        if (tid < 64) {
//            acc[tid] = acc[tid] + acc[tid + 64];
//        }
//        __syncthreads();
//    }
//}
//
//template <typename T, uint32_t kBlockSize>
//__device__ void WarpSumExpReduce(volatile T *acc, uint32_t tid) {
//    if constexpr (kBlockSize >= 64) {
//        acc[tid] = acc[tid] + acc[tid + 32];
//    }
//    if constexpr (kBlockSize >= 32) {
//        acc[tid] = acc[tid] + acc[tid + 16];
//    }
//    if constexpr (kBlockSize >= 16) {
//        acc[tid] = acc[tid] + acc[tid + 8];
//    }
//    if constexpr (kBlockSize >= 8) {
//        acc[tid] = acc[tid] + acc[tid + 4];
//    }
//    if constexpr (kBlockSize >= 4) {
//        acc[tid] = acc[tid] + acc[tid + 2];
//    }
//    if constexpr (kBlockSize >= 2) {
//        acc[tid] = acc[tid] + acc[tid + 1];
//    }
//}
//
//template <typename T, uint32_t kBlockSize>
//__global__ void MaxSumExp3(Index m, Index n, Index k, Index mk,
//                           T const *__restrict__ src, T *__restrict__ dst) {
//    // Memory model of user-maneged cache in shared memory.
//    size_t const data_size = blockDim.x * blockDim.y * blockDim.z;
//    T *cache = reinterpret_cast<T *>(extent); // Mirror of global memory.
//    // Accumulator for max-reduction and sum-reduction.
//    T *acc = reinterpret_cast<T *>(cache) + data_size;
//
//    // Obtain global and local position of the current thread.
//    auto tid = threadIdx.y;
//    auto ix = threadIdx.x + blockDim.x * blockIdx.x;
//    auto jx = threadIdx.y + blockDim.y * blockIdx.y;
//    auto kx = threadIdx.z + blockDim.z * blockIdx.z;
//    bool out_of_scope = ix >= m || jx >= k || kx >= n;
//
//    // auto it = (2 * kBlockSize) * blockIdx.y + tid;
//    // auto grid_size = (2 * kBlockSize) * gridDim.y;
//    // auto data = src + (ix + mk * kx);
//
//    // Load data from global memory to user-managed cache in shared memory.
//    if (out_of_scope) {
//        cache[tid] = -INFINITY;
//        acc[tid] = -INFINITY;
//    } else {
//        cache[tid] = src[ix + m * jx + mk * kx];
//        acc[tid] = cache[tid];
//    }
//    __syncthreads();
//
//    // Per-block max-reduction in shared memory.
//    BlockMaxReduce<T, kBlockSize>(acc, tid);
//    if (tid < 32) {
//        WarpMaxReduce<T, kBlockSize>(acc, tid);
//    }
//
//    // Per-block sumexp-reduction in shared memory.
//    T const max = acc[0];
//    acc[tid] = exp(cache[tid] - max);
//    __syncthreads();
//
//    BlockSumExpReduce<T, kBlockSize>(acc, tid);
//    if (tid < 32) {
//        WarpSumExpReduce<T, kBlockSize>(acc, tid);
//    }
//
//    // Store in global memory (output buffer) in theads from X-Z plane.
//    if (tid == 0) {
//        // Contingues tuple of (max, sum). Update accumulants in-place.
//        auto out = dst + 2 * (ix + m * kx);
//        if (auto diff = max - out[0]; diff > 0) {
//            out[0] = max;
//            out[1] = out[1] * exp(-diff) + acc[tid];
//        } else {
//            out[1] = out[1] + exp(diff) * acc[tid];
//        }
//    }
//}
//
//template <typename T> constexpr T ceil2(T value) {
//    static_assert(std::is_integral<T>::value, "integral type expected");
//    value--;
//    // Divide by 2^k for consecutive doublings of k up to 256,
//    // and then or the results.
//    value |= value >> 1;
//    value |= value >> 2;
//    value |= value >> 4;
//    if constexpr (sizeof(value) >= 2) {
//        value |= value >> 8;
//    }
//    if constexpr (sizeof(value) >= 4) {
//        value |= value >> 16;
//    }
//    if constexpr (sizeof(value) >= 8) {
//        value |= value >> 32;
//    }
//    if constexpr (sizeof(value) >= 16) {
//        value |= value >> 64;
//    }
//    if constexpr (sizeof(value) >= 32) {
//        value |= value >> 128;
//    }
//    // The result is a number of 1 bits equal to the number
//    // of bits in the original number, plus 1. That's the
//    // next highest power of 2.
//    return ++value;
//}
//
//template <typename T>
//void LaunchMaxSumExp3(hipStream_t stream, Index m, Index n, Index k,
//                      T const *src, T *dst) noexcept {
//    size_t block_size = ceil2(k);
//    if (block_size > kMaxBlockSize) {
//        block_size = kMaxBlockSize;
//    }
//
//    dim3 threads(1, block_size, 1);
//    auto noblocks = (k - 1) / threads.y + 1;
//    dim3 blocks(m, noblocks, n);
//    size_t smem = 2 * threads.x * threads.y * threads.z * sizeof(T);
//
//    if (blocks.y > 1) {
//        std::cerr << "unsupported thread block size" << std::endl;
//        std::terminate();
//    }
//
//    switch (threads.y) {
//    case 1024:
//        MaxSumExp3<T, 1024>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 512:
//        MaxSumExp3<T, 512>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 256:
//        MaxSumExp3<T, 256>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 128:
//        MaxSumExp3<T, 128>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 64:
//        MaxSumExp3<T, 64>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 32:
//        MaxSumExp3<T, 32>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 16:
//        MaxSumExp3<T, 16>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 8:
//        MaxSumExp3<T, 8>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 4:
//        MaxSumExp3<T, 4>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 2:
//        MaxSumExp3<T, 2>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    case 1:
//        MaxSumExp3<T, 1>
//            <<<blocks, threads, smem, stream>>>(m, n, k, m * k, src, dst);
//        break;
//    default:
//        std::cerr << "unsupported thread block size" << std::endl;
//        break;
//    }
//}
//
//template void LaunchMaxSumExp3<fp32_t>(hipStream_t stream, Index m, Index n,
//                                       Index k, const fp32_t *src,
//                                       fp32_t *maxsumexp) noexcept;
//
//template void LaunchMaxSumExp3<fp64_t>(hipStream_t stream, Index m, Index n,
//                                       Index k, const fp64_t *src,
//                                       fp64_t *maxsumexp) noexcept;

template <typename T>
void cuda(hipStream_t stream, Index m, Index n, Index k, const T *src,
          T *maxsumexp)
    noexcept
{
    LaunchMaxSumExp1(stream, m, n, k, src, maxsumexp);
}

template void cuda<fp32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_t *src, fp32_t *maxsumexp) noexcept;

template void cuda<fp64_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp64_t *src, fp64_t *maxsumexp) noexcept;

template void cuda<bf16_t>(hipStream_t stream, Index m, Index n, Index k,
        const bf16_t *src, bf16_t *maxsumexp) noexcept;

template void cuda<fp32_fast_tf32_t>(hipStream_t stream, Index m, Index n, Index k,
        const fp32_fast_tf32_t *src, fp32_fast_tf32_t *maxsumexp) noexcept;

} // namespace nntile::kernel::maxsumexp
